#include "hip/hip_runtime.h"

#include <cmath>
#include <vector>
#include <iostream>
#include <vector>
#include <string>
#include <algorithm>
#include <thread>
#include "ReverseQueue.cu"
#include "A5LogicalZonotopeQueue.cu"
#include <ctime>
#include ""
#include <hip/hip_runtime.h>

#pragma once

//__device__  int* d_outStream = NULL;
//__device__  int d_count = 0;

__device__   __constant__ int d_noAssumedBits = 5;


static __device__  uint8_t* GenerateLogicalZonotopeRegisterDevice(int length) {

    uint8_t* reg = new uint8_t[length];

    for (int i = 0; i < length; i++) {
        reg[i] = 2;
    }

    return reg;
}

static  __device__  uint8_t* GenerateLogicalZonotopeRegisterDevice(uint8_t* reg, int length) {
    for (int i = 0; i < length; i++) {
        reg[i] = 2;
    }

    return reg;
}

static  __device__  uint8_t* GenerateLogicalZonotopeRegisterDevice(uint8_t* reg, int length, uint8_t* point) {
    for (int i = 0; i < length - d_noAssumedBits; i++) {
        reg[i] = *point;
    }

    return reg;
}
static __device__ void FillLastNBitsWithRowDevice(uint8_t* reg, const int regLength, const uint8_t* truthTableZonotope, int rowNo, int noBits) {
    int startIndex = regLength - noBits;

    for (int i = 0; i < noBits - d_noAssumedBits; i++) {
        reg[startIndex + i] = truthTableZonotope[rowNo * noBits + i];  //TODO here we assume that we just want to point to this point
    }
}

static __device__ void PrintRegistersDevice(const LightLogicalZonotope* RA, const LightLogicalZonotope* RB, const LightLogicalZonotope* RC) {
    // Print RA
    printf("RA = {             ");
    for (int i = 0; i < RAlength; ++i) {
        if (RA[i].Generator == 0) {
            printf("%d, ", RA[i].Point);
        }
        else {
            printf("Z, ");
        }
    }
    printf("}\n");

    // Print RB
    printf("RB = {    ");
    for (int i = 0; i < RBlength; ++i) {
        if (RB[i].Generator == 0) {
            printf("%d, ", RB[i].Point);
        }
        else {
            printf("Z, ");
        }
    }
    printf("}\n");

    // Print RC
    printf("RC = { ");
    for (int i = 0; i < RClength; ++i) {
        if (RC[i].Generator == 0) {
            printf("%d, ", RC[i].Point);
        }
        else {
            printf("Z, ");
        }
    }
    printf("}\n");
}
static __device__  void PrintRegistersDevice(const uint8_t  RA[], const uint8_t  RB[], const uint8_t  RC[], int count = 0) {
    // Print RA
    printf("RA = {             ");
    for (int i = 0; i < RAlength; ++i) {
            printf("%d, ", RA[i]);
    }
    printf("}\n");

    // Print RB
    printf("RB = {    ");
    for (int i = 0; i < RBlength; ++i) {
            printf("%d, ", RB[i]);
    }
    printf("}\n");

    // Print RC
    printf("RC = { ");
    for (int i = 0; i < RClength; ++i) {
            printf("%d, ", RC[i]);
    }
    printf("}\n");
}



//static uint8_t* RA = Helper::GenerateLogicalZonotopeRegister(RAlength);
//static uint8_t* RB = Helper::GenerateLogicalZonotopeRegister(RBlength);
//static uint8_t* RC = Helper::GenerateLogicalZonotopeRegister(RClength);



//TODO this function need some cleaning of varaibles and sorting of the logic



//static int deepBitsTableLength;
//static uint8_t* threeBitsTruthTableZonotope;
//static uint8_t* sixBitsTruthTableZonotope;
//static uint8_t* deepTruthTableZonotope;
//static int RAlastZTind, RBlastZTind, RClastZTind;
//static uint8_t tempPoint = uint8_t{0, 1}; 
//static int threeBitsTableLength = 8;
//static int totalAssumedBits;
//
//static void InitializeStaticMembers();
//static  bool A5RFBZT_12_DeepModeLoop_A5Loop(uint8_t* RA, uint8_t* RB, uint8_t* RC, const std::string& indexText);
//static void A5FullKey(uint8_t* RAcurr, uint8_t* RBcurr, uint8_t* RCcurr, int initialRClastZTind, std::string indexText);
//static void finalStep(uint8_t* RAcurr, uint8_t* RBcurr, uint8_t* RCcurr, int initialRClastZTind);
//
//
//
//
//static void InitializeStaticMembers() {
//        RAlastZTind = RAlength - noAssumedBits - 1;
//        RBlastZTind = RAlength - noAssumedBits + 2;
//        RClastZTind = RAlength - noAssumedBits + 3;
//        deepBitsTableLength = static_cast<int>(std::pow(2, deepNoAssumedBits * 3));
//        threeBitsTruthTableZonotope = Helper::GetTruthTableZonotope(1 * 3);
//        sixBitsTruthTableZonotope = Helper::GetTruthTableZonotope(2 * 3);
//        deepTruthTableZonotope = Helper::GetTruthTableZonotope(deepNoAssumedBits * 3);
//
//        totalAssumedBits = deepNoAssumedBits * 3;
//    }
//
//    //for (int mainIndex = 0; mainIndex < mainIndexLength; ++mainIndex) {
//       //    // Ensure we don't create more threads than allowed
//       //    if (threads.size() >= maxThreads) {
//       //        // Join threads before creating new ones
//       //        for (auto& thread : threads) {
//       //            thread.join();
//       //        }
//       //        threads.clear();
//       //    }
//
//       //    threads.emplace_back([mainIndex, RA, RB, RC, indexText]() {
//
//    static  bool A5RFBZT_12_DeepModeLoop_A5Loop(uint8_t* RA, uint8_t* RB, uint8_t* RC, const std::string& indexText) {
//        const int mainIndexLength = 64;  // Assuming sixBitsTableLength is 64
//
//        // Set up parallel options
//        int maxThreads = std::thread::hardware_concurrency() / 4;
//        std::vector<std::thread> threads;
//
//       // int mainIndex = 8;
//
//        //// Perform parallel loop
//        //for (int mainIndex = 0; mainIndex < mainIndexLength; ++mainIndex) {
//        //    threads.emplace_back([mainIndex, RA, RB, RC, indexText]() {
//
//        // Use TBB for parallel loop
//        tbb::parallel_for(0, mainIndexLength, [&](int mainIndex) {
//       // for (int mainIndex = 0; mainIndex < mainIndexLength; ++mainIndex) {
//
//            std::string outputString;
//            bool isValid = false;
//            bool isKeyFound = false;
//
//            // Create temporary copies of RA, RB, RC
//            uint8_t* tempRA = new uint8_t[RAlength];
//            uint8_t* tempRB = new uint8_t[RBlength];
//            uint8_t* tempRC = new uint8_t[RClength];
//
//            // Copy values from RA, RB, RC to temporary arrays
//            std::copy(RA, RA + RAlength, tempRA);
//            std::copy(RB, RB + RBlength, tempRB);
//            std::copy(RC, RC + RClength, tempRC);
//
//
//            tempRA[RAlastZTind] = sixBitsTruthTableZonotope[mainIndex * 6 + 0];//[0];
//            tempRB[RBlastZTind] = sixBitsTruthTableZonotope[mainIndex * 6 + 1];//[1];
//            tempRC[RClastZTind] = sixBitsTruthTableZonotope[mainIndex * 6 + 2];//[2];
//
//            tempRA[RAlastZTind - 1] = sixBitsTruthTableZonotope[mainIndex * 6 + 3];//[3];
//            tempRB[RBlastZTind - 1] = sixBitsTruthTableZonotope[mainIndex * 6 + 4];//[4];
//            tempRC[RClastZTind - 1] = sixBitsTruthTableZonotope[mainIndex * 6 + 5];//[5];
//            if (isExactPoly)
//            {
//               // A5PolyLogicalZonotope A5PolyZonotope(tempRA, tempRB, tempRC);
//               // isValid = A5PolyZonotope.IsValidKey();
//            }
//            else
//            {
//                A5LogicalZonotopeQueue A5Zonotope(tempRA, tempRB, tempRC);
//                isValid = A5Zonotope.IsValidKey('A');
//                if (!isValid) isValid = A5Zonotope.IsValidKey('B');
//                if (!isValid) isValid = A5Zonotope.IsValidKey('C');
//                if (!isValid) isValid = A5Zonotope.IsValidKey('D');
//
//            }
//
//
//            if (isValid) {
//                outputString = "  [Deep Mode] index [" + std::to_string(mainIndex) + "] for [Main Thread] " + indexText + " is valid looking inside @ " + Helper::GetCurrentTime();
//                std::cout << outputString << std::endl;
//                A5FullKey(tempRA, tempRB, tempRC, RClastZTind - 2, indexText + " - M = " + std::to_string(mainIndex));
//                /* outputString = "  [Deep Mode] index [" + std::to_string(mainIndex) + "] for [Main Thread] " + indexText + " is FINISHED @ " + Helper::GetCurrentTime();
//                 std::cout << outputString << std::endl;*/
//            }
//            else {
//                outputString = "  [Deep Mode] index [" + std::to_string(mainIndex) + "] for [Main Thread] " + indexText + " is NOT VALID @ " + Helper::GetCurrentTime();
//                std::cout << outputString << std::endl;
//            }
//            delete[] tempRA;
//            delete[] tempRB;
//            delete[] tempRC;
//        });
//
// 
//        return false;
//    }
//
//    static void A5FullKey(uint8_t* RAcurr, uint8_t* RBcurr, uint8_t* RCcurr, int initialRClastZTind, std::string indexText) {
//        //A5PolyLogicalZonotope A5PolyZonotope(RAcurr, RBcurr, RCcurr);
//        A5LogicalZonotopeQueue A5Zonotope(RAcurr, RBcurr, RCcurr);
//        uint8_t* tempRAKey = new uint8_t[RAlength];
//        uint8_t* tempRBKey = new uint8_t[RBlength];
//        uint8_t* tempRCKey = new uint8_t[RClength];
//
//        int relativeIndex = initialRClastZTind;
//        int lastIndex = relativeIndex - noSegmants * deepNoAssumedBits;
//
//       // ReverseQueue segIndexQueue(RClength);
//        ReverseQueue segIndexQueue = initializeReverseQueue(RClength);
//
//
//        int index = 0;
//        bool isValid;
//        bool clkCondArray[4] = { false, false, false, false };
//        int deepRelativeIndex = relativeIndex - deepNoAssumedBits;
//        int deepInitialRClastZTind = deepRelativeIndex;
//        std::string outputString;
//
//        while (true) {
//            while (index < deepBitsTableLength) {
//                Helper::FillNBitsWithRow(RAcurr, deepRelativeIndex - 3, &deepTruthTableZonotope[index * totalAssumedBits], 0, deepNoAssumedBits);
//                Helper::FillNBitsWithRow(RBcurr, deepRelativeIndex, &deepTruthTableZonotope[index * totalAssumedBits], deepNoAssumedBits, deepNoAssumedBits);
//                Helper::FillNBitsWithRow(RCcurr, deepRelativeIndex + 1, &deepTruthTableZonotope[index * totalAssumedBits], deepNoAssumedBits * 2, deepNoAssumedBits);
//
//
//                isValid = false;
//                // MajorityFunction and IsValidKey functions are not provided; replace them with actual implementations.
//                if (isExactPoly)
//                {
//                   // isValid = A5PolyZonotope.IsValidKey();
//                }
//                else
//                {
//
//                    A5LogicalZonotopeQueue::MajorityFunction(RAcurr[8], RBcurr[10], RCcurr[10], clkCondArray);
//                    if (clkCondArray[0])
//                        isValid = A5Zonotope.IsValidKey('A');
//                    if (clkCondArray[1] && !isValid)
//                        isValid = A5Zonotope.IsValidKey('B');
//                    if (clkCondArray[2] && !isValid)
//                        isValid = A5Zonotope.IsValidKey('C');
//                    if (clkCondArray[3] && !isValid)
//                        isValid = A5Zonotope.IsValidKey('D');
//                }
//
//                if (isValid) {
//                    if (deepRelativeIndex == lastIndex) {
//
//                        std::copy(RAcurr, RAcurr + RAlength, tempRAKey);
//                        std::copy(RBcurr, RBcurr + RBlength, tempRBKey);
//                        std::copy(RCcurr, RCcurr + RClength, tempRCKey);
//
//                        //// Replace with actual copying logic
//                        //for (int i = 0; i < RAlength; ++i) {
//                        //    tempRAKey[i] = RAcurr[i];
//                        //}
//                        //for (int i = 0; i < RBlength; ++i) {
//                        //    tempRBKey[i] = RBcurr[i];
//                        //}
//                        //for (int i = 0; i < RClength; ++i) {
//                        //    tempRCKey[i] = RCcurr[i];
//                        //}
//
//                        finalStep(tempRAKey, tempRBKey, tempRCKey, lastIndex);
//                        /*
//                           outputString = " [Main Thread] " + indexText + " index so far is " + std::to_string(index) + " @ " + Helper::GetCurrentTime();
//                           std::cout << outputString << std::endl;*/
//
//                        index++;
//                        continue;
//                    }
//                    deepRelativeIndex = deepRelativeIndex - deepNoAssumedBits;
//                   // segIndexQueue.Enqueue(index);
//                    Enqueue(segIndexQueue, index);
//
//                    index = 0;
//                }
//                else {
//                    index++;
//                }
//            }
//
//            if (QueueSize(segIndexQueue) != 0) {
//                if (deepRelativeIndex < deepInitialRClastZTind) {
//                    for (int i = 1; i < deepNoAssumedBits + 1; i++) {
//                        RCcurr[deepRelativeIndex + i] = tempPoint;
//                        RBcurr[deepRelativeIndex - 1 + i] = tempPoint;
//                        RAcurr[deepRelativeIndex - 4 + i] = tempPoint;
//                    }
//
//                   // index = segIndexQueue.Dequeue() + 1;
//                    index = Dequeue(segIndexQueue) + 1;
//
//                    deepRelativeIndex = deepRelativeIndex + deepNoAssumedBits;
//                }
//            }
//            else {
//                outputString = " [Main Thread] " + indexText + " --- FINISHED DEEP @ " + Helper::GetCurrentTime();
//
//                std::cout << " [Main Thread] " << indexText << " --- FINISHED DEEP @ " << Helper::GetCurrentTime() << std::endl;
//                delete[] tempRAKey;
//                delete[] tempRBKey;
//                delete[] tempRCKey;
//                return;
//            }
//        }
//    }
//   
//    static void finalStep(uint8_t* RAcurr, uint8_t* RBcurr, uint8_t* RCcurr, int initialRClastZTind)
//    {
//        std::string outputString;
//        A5LogicalZonotopeQueue A5Zonotope(RAcurr, RBcurr, RCcurr);
//        //A5PolyLogicalZonotope A5PolyZonotope(RAcurr, RBcurr, RCcurr);
//
//        //ReverseQueue indexQueue(RClength);
//        ReverseQueue indexQueue = initializeReverseQueue(RClength);
//
//        int relativeIndex = initialRClastZTind;
//
//        /*  if (RAcurr[4].Point == 1 && RAcurr[5].Point == 0 && RBcurr[7].Point == 0) {
//              outputString = " [Final step]  RC[8]&[9] = " + std::to_string(RCcurr[8].Point) + ',' + std::to_string(RCcurr[9].Point) + " @ " + Helper::GetCurrentTime();
//              std::cout << outputString << std::endl;
//          }*/
//        int index = 0;
//        bool isValid;
//        bool clkCondArray[4] = { false, false, false, false };
//        while (true)
//        {
//            while (index < threeBitsTableLength)
//            {
//                // if the relativeIndex is less than 4 then we are working on RB & RC only 
//                if (relativeIndex < 4)
//                {
//                    if (relativeIndex == 0 && index > 1) // two values for RC 0 or 1 are tested now. so let's break
//                        break;
//                    if (index > 3)  // all four values for RC and RB are tested now. so let's break
//                        break;
//                }
//                else
//                    RAcurr[relativeIndex - 4] = threeBitsTruthTableZonotope[index * 3 + 0];//[0];
//                if (relativeIndex > 0)
//                    RBcurr[relativeIndex - 1] = threeBitsTruthTableZonotope[index * 3 + 1];// [1] ;
//                RCcurr[relativeIndex] = threeBitsTruthTableZonotope[index * 3 + 2];// [2] ;
//
//                isValid = false;
//            /*    if (isExactPoly)
//                {
//                    isValid = A5PolyZonotope.IsValidKey();
//                }
//                else
//                {*/
//
//                    A5LogicalZonotopeQueue::MajorityFunction(RAcurr[8], RBcurr[10], RCcurr[10], clkCondArray);
//                    if (clkCondArray[0])
//                        isValid = A5Zonotope.IsValidKey('A');
//                    if (clkCondArray[1] && !isValid)
//                        isValid = A5Zonotope.IsValidKey('B');
//                    if (clkCondArray[2] && !isValid)
//                        isValid = A5Zonotope.IsValidKey('C');
//                    if (clkCondArray[3] && !isValid)
//                        isValid = A5Zonotope.IsValidKey('D');
//               // }
//                if (isValid)
//                {
//                    if (relativeIndex == 0)
//                    {
//                        std::cout << "\n*** We found a Key @ " << Helper::GetCurrentTime() << "***\n";
//                        Helper::PrintRegisters(RAcurr, RBcurr, RCcurr);
//                        return;
//                    }
//                    relativeIndex--;
//                   // indexQueue.Enqueue(index);
//                    Enqueue(indexQueue, index);
//                    index = 0;
//                }
//                else
//                    index++;
//            }
//
//            if (QueueSize(indexQueue) != 0)
//            {
//                if (relativeIndex < initialRClastZTind)
//                {
//                    RCcurr[relativeIndex] = tempPoint;
//                   // index = indexQueue.Dequeue() + 1;
//                    index = Dequeue(indexQueue) + 1;
//
//                    if (relativeIndex > 0)
//                    {
//                        RBcurr[relativeIndex - 1] = tempPoint;
//                        if (relativeIndex > 3)
//                            RAcurr[relativeIndex - 4] = tempPoint;
//                    }
//                    relativeIndex++;
//                }
//            }
//            else
//            {
//                return;
//            }
//        }
//    }

