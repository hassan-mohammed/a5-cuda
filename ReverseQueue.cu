#define ReverseQueueSize 23

#include <iostream>
#include <crt/host_defines.h>
#pragma once
#include <hip/hip_runtime.h>


struct ReverseQueue {
    int array[ReverseQueueSize];
    size_t head;      // The index from which to dequeue if the queue isn't empty.
    size_t size;      // Number of elements currently in the queue.
    size_t capacity = ReverseQueueSize;  // Capacity of the array.
};

// Initialize ReverseQueue
static __device__ __host__ void initializeReverseQueue(ReverseQueue * queue) {
    queue->head = 0;
    queue->size = 0;

}

static __device__ __host__ size_t QueueSize(const ReverseQueue& queue) {
    return queue.size;
}

static __device__ __host__ size_t QueueCapacity(const ReverseQueue& queue) {
    return queue.capacity;
}

static __device__ __host__ void Enqueue(ReverseQueue& queue, int element) {
    //TODO:Conv throw exception isn't allowed    if (queue.size == queue.capacity) throw "Queue is full";

    queue.array[queue.head] = element;
    queue.head++;
    queue.size++;
}

static __device__ __host__ int Dequeue(ReverseQueue& queue) {
    //TODO:Conv throw exception isn't allowed    if (queue.size == 0) throw "Empty queue";

    int removed = queue.array[queue.head - 1];
    queue.head--;
    queue.size--;
    queue.array[queue.head] = 0;  // Assuming 0 is the default value for an empty slot
    return removed;
}
