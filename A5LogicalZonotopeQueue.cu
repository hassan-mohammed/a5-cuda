#include "hip/hip_runtime.h"
#define MAX_STACK_SIZE 64
#define CountConst 64


#include <iostream>
#include <array>
#include <vector>
#include <unordered_map>
#include "Helper.cpp"
#include ""
#include <hip/hip_runtime.h>
#pragma once

__device__  __constant__ const int d_RAlength = 19;
__device__  __constant__ const int d_RBlength = 22;
__device__  __constant__ const int d_RClength = 23;

struct StackItem {
    int  countCurr;
    int  RAind;
    int  RBind;
    int  RCind;
    bool clkCondArray[4];
};

struct A5LogicalZonotopeQueue {

    int countCurr;

    LightLogicalZonotope* _RAMain;
    LightLogicalZonotope* _RBMain;
    LightLogicalZonotope* _RCMain;
    bool* clkCondArray;

    LightLogicalZonotope _tempRA[d_RAlength + CountConst];
    LightLogicalZonotope _tempRB[d_RAlength + CountConst];
    LightLogicalZonotope _tempRC[d_RClength + CountConst];

    StackItem _stack[MAX_STACK_SIZE];
    StackItem* dctItemPointer;

    int stackIndex;

    bool clkCondA = false, clkCondB = false, clkCondC = false, clkCondD = false;



};


static __device__ __host__  void XorDevice(const LightLogicalZonotope& zonotope1, const LightLogicalZonotope& zonotope2, const LightLogicalZonotope& zonotope3, LightLogicalZonotope& zonotopeOut) {
    if (zonotope3.Generator != 0)
        zonotopeOut.Generator = 1;
    else if (zonotope2.Generator != 0)
        zonotopeOut.Generator = 1;
    else if (zonotope1.Generator != 0)
        zonotopeOut.Generator = 1;
    else {
        zonotopeOut.Generator = 0;
        zonotopeOut.Point = zonotope1.Point ^ zonotope2.Point ^ zonotope3.Point;
    }
}

__device__ __host__ static bool IsValidKey(A5LogicalZonotopeQueue& A5LZQueue, char clkCond);
__device__ __host__ static void ClkRegistersNew(A5LogicalZonotopeQueue& A5LZQueue, char clkCond, int& RAind, int& RBind, int& RCind) ;
__device__ __host__ static void MajorityFunction(A5LogicalZonotopeQueue& A5LZQueue, int& RAind, int& RBind, int& RCind)                 ;
__device__ __host__ static void AddNewItem(A5LogicalZonotopeQueue& A5LZQueue, int& countCurr, int&  RAind, int& RBind, int& RCind)        ;


static __device__   LightLogicalZonotope uncertainPoint = LightLogicalZonotope{ 0, 1 };
static __device__   LightLogicalZonotope oneCertainPoint = LightLogicalZonotope{ 1, 0 };
static __device__   LightLogicalZonotope zeroCertainPoint = LightLogicalZonotope{ 0, 0 };
const static __device__    int* _outStream;
static __device__   int _count;



    // Assuming FindA5KeyLightZT class is defined appropriately
__device__ __host__ static void InitializeA5LogicalZonotopeQueue(A5LogicalZonotopeQueue* A5LZQueue, LightLogicalZonotope* RA, LightLogicalZonotope* RB, LightLogicalZonotope* RC, const int* outStream, int count) {
        _outStream = outStream;
       _count = count;
       
       A5LZQueue->_RAMain = RA;
       A5LZQueue->_RBMain = RB;
       A5LZQueue->_RCMain = RC;



    }

   
__device__ __host__  static bool IsValidKey(A5LogicalZonotopeQueue& A5LZQueue, char clkCond) {
        A5LZQueue.stackIndex = 0;
        int countCurr = _count;
        LightLogicalZonotope  outA;


        int RAind, RBind, RCind;

        RAind = RBind = RCind = _count;

        //TODO is there's a better and faster way to do that instead of for loops
        /*std::copy(A5LZQueue._RAMain, A5LZQueue._RAMain + RAlength, A5LZQueue._tempRA + _count);
        std::copy(A5LZQueue._RBMain, A5LZQueue._RBMain + RBlength, A5LZQueue._tempRB + _count);
        std::copy(A5LZQueue._RCMain, A5LZQueue._RCMain + RClength, A5LZQueue._tempRC + _count);*/
        int i = 0;

        for (; i < d_RAlength; ++i) {
            A5LZQueue._tempRA[i + _count] = A5LZQueue._RAMain[i];
            A5LZQueue._tempRB[i + _count] = A5LZQueue._RBMain[i];
            A5LZQueue._tempRC[i + _count] = A5LZQueue._RCMain[i];
        }

        for (i = d_RAlength; i < d_RBlength; ++i) {
            A5LZQueue._tempRB[i + _count] = A5LZQueue._RBMain[i];
            A5LZQueue._tempRC[i + _count] = A5LZQueue._RCMain[i];
        }

        for (i = d_RBlength; i < d_RClength; ++i) {
            A5LZQueue._tempRC[i + _count] = A5LZQueue._RCMain[i];
        }
        ///////////// first round ////////////

        // calculating the output
        outA.Generator = 1;
        XorDevice(A5LZQueue._tempRA[RAind + 18], A5LZQueue._tempRB[RBind + 21], A5LZQueue._tempRC[RCind + 22], outA);
        if (!(outA.Point == _outStream[_count - countCurr]))
            return false;

        // calculating the feedback bits
        //CalculateFeedbackBits();
        // Clock the registers


        ClkRegistersNew(A5LZQueue, clkCond, RAind, RBind, RCind);

        //Helper.PrintRegisters( A5LZQueue._tempRA,  A5LZQueue._tempRB,  A5LZQueue._tempRC);

        ///////////// second round ////////////
        countCurr--;
        // calculating the output
        XorDevice(A5LZQueue._tempRA[RAind + 18], A5LZQueue._tempRB[RBind + 21], A5LZQueue._tempRC[RCind + 22], outA);
        if (!(outA.Generator != 0 || outA.Point == _outStream[_count - countCurr]))
            return false;

        // calculate the majority bits for all registers using majority function
        //clkCondArray = MajorityFunction();

        AddNewItem(A5LZQueue, countCurr, RAind, RBind, RCind);
        // clkCondArray = _stack[stackIndex].Item5;
        while (countCurr > 1) {
            countCurr--;
            A5LZQueue.clkCondArray = &A5LZQueue._stack[A5LZQueue.stackIndex].clkCondArray[0];
            if (A5LZQueue.clkCondArray[0] == true) {
                A5LZQueue.clkCondArray[0] = false;
                clkCond = 'A';
                XorDevice(A5LZQueue._tempRA[RAind + 17], A5LZQueue._tempRB[RBind + 20], A5LZQueue._tempRC[RCind + 21], outA);
                if (outA.Generator != 0 || outA.Point == _outStream[_count - countCurr]) {
                    ClkRegistersNew(A5LZQueue, clkCond, RAind, RBind, RCind);
                    AddNewItem(A5LZQueue, countCurr, RAind, RBind, RCind);
                    continue;
                }
            }

            if (A5LZQueue.clkCondArray[1] == true) {
                A5LZQueue.clkCondArray[1] = false;
                clkCond = 'B';
                XorDevice(A5LZQueue._tempRA[RAind + 17], A5LZQueue._tempRB[RBind + 20], A5LZQueue._tempRC[RCind + 22], outA);
                if (outA.Generator != 0 || outA.Point == _outStream[_count - countCurr]) {
                    ClkRegistersNew(A5LZQueue, clkCond, RAind, RBind, RCind);
                    AddNewItem(A5LZQueue, countCurr, RAind, RBind, RCind);
                    continue;
                }
            }
            if (A5LZQueue.clkCondArray[2] == true) {
                A5LZQueue.clkCondArray[2] = false;
                clkCond = 'C';

                XorDevice(A5LZQueue._tempRA[RAind + 17], A5LZQueue._tempRB[RBind + 21], A5LZQueue._tempRC[RCind + 21], outA);

                if (outA.Generator != 0 || outA.Point == _outStream[_count - countCurr]) {
                    ClkRegistersNew(A5LZQueue, clkCond, RAind, RBind, RCind);
                    AddNewItem(A5LZQueue, countCurr, RAind, RBind, RCind);
                    continue;
                }
            }
            if (A5LZQueue.clkCondArray[3] == true) {
                A5LZQueue.clkCondArray[3] = false;
                clkCond = 'D';

                XorDevice(A5LZQueue._tempRA[RAind + 18], A5LZQueue._tempRB[RBind + 20], A5LZQueue._tempRC[RCind + 21], outA);
                if (outA.Generator != 0 || outA.Point == _outStream[_count - countCurr]) {
                    ClkRegistersNew(A5LZQueue, clkCond, RAind, RBind, RCind);
                    AddNewItem(A5LZQueue, countCurr, RAind, RBind, RCind);
                    continue;

                }
            }
            A5LZQueue.stackIndex--;
            if (A5LZQueue.stackIndex == 0) {
                return false;
            }
            // retrieve elements from the stack at key =  A5LZQueue.stackIndex
           // std::tie(RAind, RBind, RCind, countCurr, A5LZQueue.clkCondArray) = A5LZQueue._stack[A5LZQueue.stackIndex];

            //StackItem& stackItem = A5LZQueue._stack[A5LZQueue.stackIndex];
            RAind = A5LZQueue._stack[A5LZQueue.stackIndex].RAind;
            RBind = A5LZQueue._stack[A5LZQueue.stackIndex].RBind;
            RCind = A5LZQueue._stack[A5LZQueue.stackIndex].RCind;
            countCurr = A5LZQueue._stack[A5LZQueue.stackIndex].countCurr;
            A5LZQueue.clkCondArray = &A5LZQueue._stack[A5LZQueue.stackIndex].clkCondArray[0];
        }

        return true;
    }

__device__ __host__ __inline__ static void ClkRegistersNew(A5LogicalZonotopeQueue& A5LZQueue, char clkCond, int& RAind, int& RBind, int& RCind) {
        int FeedbackPoint;
        if (clkCond == 'A' || clkCond == 'B' || clkCond == 'C') {
            if (A5LZQueue._tempRA[RAind + 13].Generator == 0 && A5LZQueue._tempRA[RAind + 16].Generator == 0 &&
                A5LZQueue._tempRA[RAind + 17].Generator == 0 && A5LZQueue._tempRA[RAind + 18].Generator == 0) {
                FeedbackPoint = A5LZQueue._tempRA[RAind + 13].Point ^ A5LZQueue._tempRA[RAind + 16].Point ^
                    A5LZQueue._tempRA[RAind + 17].Point ^ A5LZQueue._tempRA[RAind + 18].Point;
                // std::copy( A5LZQueue._tempRA + 1,  A5LZQueue._tempRA + RAlength,  A5LZQueue._tempRA);
                A5LZQueue._tempRA[RAind - 1] = (FeedbackPoint == 1) ? oneCertainPoint : zeroCertainPoint;
            }
            else {
                // std::copy( A5LZQueue._tempRA + 1,  A5LZQueue._tempRA + RAlength,  A5LZQueue._tempRA);
                A5LZQueue._tempRA[RAind - 1] = uncertainPoint;
            }
            RAind--;
        }

        if (clkCond == 'A' || clkCond == 'B' || clkCond == 'D') {
            if (A5LZQueue._tempRB[RBind + 20].Generator == 0 && A5LZQueue._tempRB[RBind + 21].Generator == 0) {
                FeedbackPoint = A5LZQueue._tempRB[RBind + 20].Point ^ A5LZQueue._tempRB[RBind + 21].Point;
                // std::copy( A5LZQueue._tempRB + 1,  A5LZQueue._tempRB + RBlength,  A5LZQueue._tempRB);
                A5LZQueue._tempRB[RBind - 1] = (FeedbackPoint == 1) ? oneCertainPoint : zeroCertainPoint;
            }
            else {
                // std::copy( A5LZQueue._tempRB + 1,  A5LZQueue._tempRB + RBlength,  A5LZQueue._tempRB);
                A5LZQueue._tempRB[RBind - 1] = uncertainPoint;
            }
            RBind--;
        }

        if (clkCond == 'A' || clkCond == 'C' || clkCond == 'D') {
            if (A5LZQueue._tempRC[RCind + 7].Generator == 0 && A5LZQueue._tempRC[RCind + 20].Generator == 0 &&
                A5LZQueue._tempRC[RCind + 21].Generator == 0 && A5LZQueue._tempRC[RCind + 22].Generator == 0) {
                FeedbackPoint = A5LZQueue._tempRC[RCind + 7].Point ^ A5LZQueue._tempRC[RCind + 20].Point ^
                    A5LZQueue._tempRC[RCind + 21].Point ^ A5LZQueue._tempRC[RCind + 22].Point;
                // std::copy( A5LZQueue._tempRC + 1,  A5LZQueue._tempRC + RClength,  A5LZQueue._tempRC);
                A5LZQueue._tempRC[RCind - 1] = (FeedbackPoint == 1) ? oneCertainPoint : zeroCertainPoint;
            }
            else {
                // std::copy( A5LZQueue._tempRC + 1,  A5LZQueue._tempRC + RClength,  A5LZQueue._tempRC);
                A5LZQueue._tempRC[RCind - 1] = uncertainPoint;
            }
            RCind--;
        }
    }

__device__ __host__ static void MajorityFunction(A5LogicalZonotopeQueue& A5LZQueue, int& RAind, int& RBind, int& RCind) {
        LightLogicalZonotope _RA8 = A5LZQueue._tempRA[RAind + 8];
        LightLogicalZonotope _RB10 = A5LZQueue._tempRB[RBind + 10];
        LightLogicalZonotope _RC10 = A5LZQueue._tempRC[RCind + 10];

         A5LZQueue.clkCondA = false;
         A5LZQueue.clkCondB = false;
         A5LZQueue.clkCondC = false;
         A5LZQueue.clkCondD = false;

        // Check if RA9, RB11, RC11 generator property is empty or not
        if (_RA8.Generator == 0 && _RB10.Generator == 0 && _RC10.Generator == 0) {
            // If empty, then check if RA9, RB11, RC11 points are equal or not
            if (_RA8.Point == _RB10.Point && _RB10.Point == _RC10.Point) {
                A5LZQueue.clkCondA = true;
            }
            else if (_RA8.Point == _RB10.Point) {
                A5LZQueue.clkCondB = true;
            }
            else if (_RA8.Point == _RC10.Point) {
                A5LZQueue.clkCondC = true;
            }
            else if (_RB10.Point == _RC10.Point) {
                A5LZQueue.clkCondD = true;
            }
        }
        else if (_RA8.Generator == 0 && _RB10.Generator == 0) {
            if (_RA8.Point == _RB10.Point) {
                A5LZQueue.clkCondA = true;
                A5LZQueue.clkCondB = true;
            }
            else {
                A5LZQueue.clkCondC = true;
                A5LZQueue.clkCondD = true;
            }
        }
        else if (_RA8.Generator == 0 && _RC10.Generator == 0) {
            if (_RA8.Point == _RC10.Point) {
                A5LZQueue.clkCondA = true;
                A5LZQueue.clkCondC = true;
            }
            else {
                A5LZQueue.clkCondB = true;
                A5LZQueue.clkCondD = true;
            }
        }
        // Check the RC11 and RB11 generators
        else if (_RB10.Generator == 0 && _RC10.Generator == 0) {
            if (_RB10.Point == _RC10.Point) {
                A5LZQueue.clkCondA = true;
                A5LZQueue.clkCondD = true;
            }
            else {
                A5LZQueue.clkCondB = true;
                A5LZQueue.clkCondC = true;
            }
        }
        else {
            A5LZQueue.clkCondA = true;
            A5LZQueue.clkCondB = true;
            A5LZQueue.clkCondC = true;
            A5LZQueue.clkCondD = true;
        }
        A5LZQueue.clkCondArray = &A5LZQueue.dctItemPointer->clkCondArray[0];

        A5LZQueue.clkCondArray[0] = A5LZQueue.clkCondA;
        A5LZQueue.clkCondArray[1] = A5LZQueue.clkCondB;
        A5LZQueue.clkCondArray[2] = A5LZQueue.clkCondC;
        A5LZQueue.clkCondArray[3] = A5LZQueue.clkCondD;
    }



static __device__ __host__  void AddNewItem(A5LogicalZonotopeQueue& A5LZQueue, int& countCurr,  int& RAind, int& RBind, int& RCind) {
        A5LZQueue.stackIndex++;
        // A5LZQueue.dctItem = A5LZQueue._stack[A5LZQueue.stackIndex];
        A5LZQueue.dctItemPointer = &A5LZQueue._stack[A5LZQueue.stackIndex];

        A5LZQueue.dctItemPointer->countCurr = countCurr;
        A5LZQueue.dctItemPointer->RAind = RAind;
        A5LZQueue.dctItemPointer->RBind = RBind;
        A5LZQueue.dctItemPointer->RCind = RCind;

        //  A5LZQueue.clkCondArray = A5LZQueue.dctItem.clkCondArray;
        MajorityFunction(A5LZQueue, RAind, RBind, RCind);
    }



__device__ __host__ static void MajorityFunction(LightLogicalZonotope& RA8, LightLogicalZonotope& RB10, LightLogicalZonotope& RC10, bool(&clkCondArray)[4])
{
    clkCondArray[0] = false;
    clkCondArray[1] = false;
    clkCondArray[2] = false;
    clkCondArray[3] = false;

    // Check if RA9, RB11, RC11 generator property is empty or not
    if (RA8.Generator == 0 && RB10.Generator == 0 && RC10.Generator == 0)
    {
        // If empty, then check if RA9, RB11, RC11 points are equal or not
        if (RA8.Point == RB10.Point && RB10.Point == RC10.Point) {
            clkCondArray[0] = true;
        }
        else if (RA8.Point == RB10.Point) {
            clkCondArray[1] = true;
        }
        else if (RA8.Point == RC10.Point) {
            clkCondArray[2] = true;
        }
        else if (RB10.Point == RC10.Point) {
            clkCondArray[3] = true;
        }
    }
    else if (RA8.Generator == 0 && RB10.Generator == 0) {
        if (RA8.Point == RB10.Point) {
            clkCondArray[0] = true;
            clkCondArray[1] = true;
        }
        else {
            clkCondArray[2] = true;
            clkCondArray[3] = true;
        }
    }
    else if (RA8.Generator == 0 && RC10.Generator == 0) {
        if (RA8.Point == RC10.Point) {
            clkCondArray[0] = true;
            clkCondArray[2] = true;
        }
        else {
            clkCondArray[1] = true;
            clkCondArray[3] = true;
        }
    }
    // Check the RC11 and RB11 generators
    else if (RB10.Generator == 0 && RC10.Generator == 0) {
        if (RB10.Point == RC10.Point) {
            clkCondArray[0] = true;
            clkCondArray[3] = true;
        }
        else {
            clkCondArray[1] = true;
            clkCondArray[2] = true;
        }
    }
    else {
        clkCondArray[0] = true;
        clkCondArray[1] = true;
        clkCondArray[2] = true;
        clkCondArray[3] = true;
    }



}

    // TODO: create a Destructor for the  struct
    //~A5LogicalZonotopeQueue() {
    //    for (int i = 0; i < _count; i++) {
    //        delete[] std::get<4>(_stack[i]);  // Free the bool array
    //    }
    //    delete[]  A5LZQueue._tempRA;  // Free the LightLogicalZonotope arrays
    //    delete[]  A5LZQueue._tempRB;
    //    delete[]  A5LZQueue._tempRC;
    // 
    // 
    //}



