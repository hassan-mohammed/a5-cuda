#include "hip/hip_runtime.h"
#define MAX_STACK_SIZE 100
#define _count 64


#include <iostream>
#include <array>
#include <vector>
#include <unordered_map>
#include "Helper.cpp"
#include ""
#include <hip/hip_runtime.h>
#pragma once

__device__  __constant__ const int d_RAlength = 19;
__device__  __constant__ const int d_RBlength = 22;
__device__  __constant__ const int d_RClength = 23;

struct StackItem {
    int  countCurr;
    int  RAind;
    int  RBind;
    int  RCind;
    bool clkCondArray[4];
};

struct A5LogicalZonotopeQueue {

    int countCurr;

    uint8_t* _RAMain;
    uint8_t* _RBMain;
    uint8_t* _RCMain;
    bool* clkCondArray;

    uint8_t _tempRA[d_RAlength + _count];
    uint8_t _tempRB[d_RAlength + _count];
    uint8_t _tempRC[d_RClength + _count];

    StackItem _stack[MAX_STACK_SIZE];
    StackItem* dctItemPointer;

    int stackIndex;

    bool clkCondA = false, clkCondB = false, clkCondC = false, clkCondD = false;



};


static __device__ __host__ void XorDevice(const uint8_t& zonotope1, const uint8_t& zonotope2, const uint8_t& zonotope3, uint8_t& zonotopeOut) {
    if (zonotope2 == 2 || zonotope3 == 2 || zonotope1 == 2)
        zonotopeOut = 2;

    else {

        zonotopeOut = zonotope1 ^ zonotope2 ^ zonotope3;
    }

}

__device__ __host__ static bool IsValidKey(A5LogicalZonotopeQueue& A5LZQueue, char clkCond);
__device__ __host__ static void ClkRegistersNew(A5LogicalZonotopeQueue& A5LZQueue, char clkCond, int& RAind, int& RBind, int& RCind) ;
__device__ __host__ static void MajorityFunction(A5LogicalZonotopeQueue& A5LZQueue, int& RAind, int& RBind, int& RCind)                 ;
__device__ __host__ static void AddNewItem(A5LogicalZonotopeQueue& A5LZQueue, int& countCurr, int&  RAind, int& RBind, int& RCind)        ;


__device__ __constant__ const  uint8_t _outStream[_count] = { 1,0,0,0,0,1,1,0,1,0,0,0,0,0,0,1,0,0,0,0,1,1,0,1,0,0,1,1,0,1,1,0,1,0,1,1,0,0,0,0,0,0,0,0,1,0,0,0,1,0,0,0,0,0,1,0,1,1,0,0,0,0,0,1 };




    // Assuming FindA5KeyLightZT class is defined appropriately
__device__ __host__ static void InitializeA5LogicalZonotopeQueue(A5LogicalZonotopeQueue* A5LZQueue, uint8_t* RA, uint8_t* RB, uint8_t* RC, const int* outStream, int count) {
       
       A5LZQueue->_RAMain = RA;
       A5LZQueue->_RBMain = RB;
       A5LZQueue->_RCMain = RC;



    }

   
__device__ __host__  static bool IsValidKey(A5LogicalZonotopeQueue& A5LZQueue, char clkCond) {
        A5LZQueue.stackIndex = 0;
        int countCurr = _count;
        uint8_t  outA;


        int RAind, RBind, RCind;

        RAind = RBind = RCind = _count;

        //TODO is there's a better and faster way to do that instead of for loops
        /*std::copy(A5LZQueue._RAMain, A5LZQueue._RAMain + RAlength, A5LZQueue._tempRA + _count);
        std::copy(A5LZQueue._RBMain, A5LZQueue._RBMain + RBlength, A5LZQueue._tempRB + _count);
        std::copy(A5LZQueue._RCMain, A5LZQueue._RCMain + RClength, A5LZQueue._tempRC + _count);*/
        int i = 0;

        for (; i < d_RAlength; ++i) {
            A5LZQueue._tempRA[i + _count] = A5LZQueue._RAMain[i];
            A5LZQueue._tempRB[i + _count] = A5LZQueue._RBMain[i];
            A5LZQueue._tempRC[i + _count] = A5LZQueue._RCMain[i];
        }

        for (i = d_RAlength; i < d_RBlength; ++i) {
            A5LZQueue._tempRB[i + _count] = A5LZQueue._RBMain[i];
            A5LZQueue._tempRC[i + _count] = A5LZQueue._RCMain[i];
        }

        for (i = d_RBlength; i < d_RClength; ++i) {
            A5LZQueue._tempRC[i + _count] = A5LZQueue._RCMain[i];
        }
        ///////////// first round ////////////

        // calculating the output
        outA = 2;
        XorDevice(A5LZQueue._tempRA[RAind + 18], A5LZQueue._tempRB[RBind + 21], A5LZQueue._tempRC[RCind + 22], outA);
        if (!(outA == _outStream[0]))
            return false;

        // calculating the feedback bits
        //CalculateFeedbackBits();
        // Clock the registers


        ClkRegistersNew(A5LZQueue, clkCond, RAind, RBind, RCind);

        //Helper.PrintRegisters( A5LZQueue._tempRA,  A5LZQueue._tempRB,  A5LZQueue._tempRC);

        ///////////// second round ////////////
        countCurr--;
        // calculating the output
        XorDevice(A5LZQueue._tempRA[RAind + 18], A5LZQueue._tempRB[RBind + 21], A5LZQueue._tempRC[RCind + 22], outA);
        if (!( outA == _outStream[_count - countCurr]))
            return false;

        // calculate the majority bits for all registers using majority function
        //clkCondArray = MajorityFunction();

        AddNewItem(A5LZQueue, countCurr, RAind, RBind, RCind);
        // clkCondArray = _stack[stackIndex].Item5;
        while (countCurr > 1) {
            countCurr--;
            A5LZQueue.clkCondArray = &A5LZQueue._stack[A5LZQueue.stackIndex].clkCondArray[0];
            if (A5LZQueue.clkCondArray[0] == true) {
                A5LZQueue.clkCondArray[0] = false;
                clkCond = 'A';
                XorDevice(A5LZQueue._tempRA[RAind + 17], A5LZQueue._tempRB[RBind + 20], A5LZQueue._tempRC[RCind + 21], outA);
                if (outA== 2  || outA == _outStream[_count - countCurr]) {
                    ClkRegistersNew(A5LZQueue, clkCond, RAind, RBind, RCind);
                    AddNewItem(A5LZQueue, countCurr, RAind, RBind, RCind);
                    continue;
                }
            }

            if (A5LZQueue.clkCondArray[1] == true) {
                A5LZQueue.clkCondArray[1] = false;
                clkCond = 'B';
                XorDevice(A5LZQueue._tempRA[RAind + 17], A5LZQueue._tempRB[RBind + 20], A5LZQueue._tempRC[RCind + 22], outA);
                if (outA== 2  || outA == _outStream[_count - countCurr]) {
                    ClkRegistersNew(A5LZQueue, clkCond, RAind, RBind, RCind);
                    AddNewItem(A5LZQueue, countCurr, RAind, RBind, RCind);
                    continue;
                }
            }
            if (A5LZQueue.clkCondArray[2] == true) {
                A5LZQueue.clkCondArray[2] = false;
                clkCond = 'C';

                XorDevice(A5LZQueue._tempRA[RAind + 17], A5LZQueue._tempRB[RBind + 21], A5LZQueue._tempRC[RCind + 21], outA);

                if (outA== 2  || outA == _outStream[_count - countCurr]) {
                    ClkRegistersNew(A5LZQueue, clkCond, RAind, RBind, RCind);
                    AddNewItem(A5LZQueue, countCurr, RAind, RBind, RCind);
                    continue;
                }
            }
            if (A5LZQueue.clkCondArray[3] == true) {
                A5LZQueue.clkCondArray[3] = false;
                clkCond = 'D';

                XorDevice(A5LZQueue._tempRA[RAind + 18], A5LZQueue._tempRB[RBind + 20], A5LZQueue._tempRC[RCind + 21], outA);
                if (outA== 2  || outA == _outStream[_count - countCurr]) {
                    ClkRegistersNew(A5LZQueue, clkCond, RAind, RBind, RCind);
                    AddNewItem(A5LZQueue, countCurr, RAind, RBind, RCind);
                    continue;

                }
            }
            A5LZQueue.stackIndex--;
            if (A5LZQueue.stackIndex == 0) {
                return false;
            }
            // retrieve elements from the stack at key =  A5LZQueue.stackIndex
           // std::tie(RAind, RBind, RCind, countCurr, A5LZQueue.clkCondArray) = A5LZQueue._stack[A5LZQueue.stackIndex];

            //StackItem& stackItem = A5LZQueue._stack[A5LZQueue.stackIndex];
            RAind = A5LZQueue._stack[A5LZQueue.stackIndex].RAind;
            RBind = A5LZQueue._stack[A5LZQueue.stackIndex].RBind;
            RCind = A5LZQueue._stack[A5LZQueue.stackIndex].RCind;
            countCurr = A5LZQueue._stack[A5LZQueue.stackIndex].countCurr;
            A5LZQueue.clkCondArray = &A5LZQueue._stack[A5LZQueue.stackIndex].clkCondArray[0];
        }

        return true;
    }

__device__ __host__ static void ClkRegistersNew(A5LogicalZonotopeQueue& A5LZQueue, char clkCond, int& RAind, int& RBind, int& RCind) {
    uint8_t  FeedbackPoint;
        if (clkCond == 'A' || clkCond == 'B' || clkCond == 'C') {
            if (A5LZQueue._tempRA[RAind + 13]!=2 && A5LZQueue._tempRA[RAind + 16]!=2 &&
                A5LZQueue._tempRA[RAind + 17]!=2 && A5LZQueue._tempRA[RAind + 18]!=2) {
                FeedbackPoint = A5LZQueue._tempRA[RAind + 13] ^ A5LZQueue._tempRA[RAind + 16] ^
                    A5LZQueue._tempRA[RAind + 17] ^ A5LZQueue._tempRA[RAind + 18];
                // std::copy( A5LZQueue._tempRA + 1,  A5LZQueue._tempRA + RAlength,  A5LZQueue._tempRA);
                A5LZQueue._tempRA[RAind - 1] = FeedbackPoint;
            }
            else {
                // std::copy( A5LZQueue._tempRA + 1,  A5LZQueue._tempRA + RAlength,  A5LZQueue._tempRA);
                A5LZQueue._tempRA[RAind - 1] = 2;
            }
            RAind--;
        }

        if (clkCond == 'A' || clkCond == 'B' || clkCond == 'D') {
            if (A5LZQueue._tempRB[RBind + 20]!=2 && A5LZQueue._tempRB[RBind + 21]!=2) {
                FeedbackPoint = A5LZQueue._tempRB[RBind + 20] ^ A5LZQueue._tempRB[RBind + 21];
                // std::copy( A5LZQueue._tempRB + 1,  A5LZQueue._tempRB + RBlength,  A5LZQueue._tempRB);
                A5LZQueue._tempRB[RBind - 1] = FeedbackPoint;
            }
            else {
                // std::copy( A5LZQueue._tempRB + 1,  A5LZQueue._tempRB + RBlength,  A5LZQueue._tempRB);
                A5LZQueue._tempRB[RBind - 1] = 2;
            }
            RBind--;
        }

        if (clkCond == 'A' || clkCond == 'C' || clkCond == 'D') {
            if (A5LZQueue._tempRC[RCind + 7]!=2 && A5LZQueue._tempRC[RCind + 20]!=2 &&
                A5LZQueue._tempRC[RCind + 21]!=2 && A5LZQueue._tempRC[RCind + 22]!=2) {
                FeedbackPoint = A5LZQueue._tempRC[RCind + 7] ^ A5LZQueue._tempRC[RCind + 20] ^
                    A5LZQueue._tempRC[RCind + 21] ^ A5LZQueue._tempRC[RCind + 22];
                // std::copy( A5LZQueue._tempRC + 1,  A5LZQueue._tempRC + RClength,  A5LZQueue._tempRC);
                A5LZQueue._tempRC[RCind - 1] = FeedbackPoint;
            }
            else {
                // std::copy( A5LZQueue._tempRC + 1,  A5LZQueue._tempRC + RClength,  A5LZQueue._tempRC);
                A5LZQueue._tempRC[RCind - 1] = 2;
            }
            RCind--;
        }
    }

__device__ __host__ static void MajorityFunction(A5LogicalZonotopeQueue& A5LZQueue, int& RAind, int& RBind, int& RCind) {
        uint8_t _RA8 = A5LZQueue._tempRA[RAind + 8];
        uint8_t _RB10 = A5LZQueue._tempRB[RBind + 10];
        uint8_t _RC10 = A5LZQueue._tempRC[RCind + 10];

         A5LZQueue.clkCondA = false;
         A5LZQueue.clkCondB = false;
         A5LZQueue.clkCondC = false;
         A5LZQueue.clkCondD = false;

        // Check if RA9, RB11, RC11 generator property is empty or not
        if (_RA8!=2 && _RB10!=2 && _RC10!=2) {
            // If empty, then check if RA9, RB11, RC11 points are equal or not
            if (_RA8 == _RB10 && _RB10 == _RC10) {
                A5LZQueue.clkCondA = true;
            }
            else if (_RA8 == _RB10) {
                A5LZQueue.clkCondB = true;
            }
            else if (_RA8 == _RC10) {
                A5LZQueue.clkCondC = true;
            }
            else if (_RB10 == _RC10) {
                A5LZQueue.clkCondD = true;
            }
        }
        else if (_RA8!=2 && _RB10!=2) {
            if (_RA8 == _RB10) {
                A5LZQueue.clkCondA = true;
                A5LZQueue.clkCondB = true;
            }
            else {
                A5LZQueue.clkCondC = true;
                A5LZQueue.clkCondD = true;
            }
        }
        else if (_RA8!=2 && _RC10!=2) {
            if (_RA8 == _RC10) {
                A5LZQueue.clkCondA = true;
                A5LZQueue.clkCondC = true;
            }
            else {
                A5LZQueue.clkCondB = true;
                A5LZQueue.clkCondD = true;
            }
        }
        // Check the RC11 and RB11 generators
        else if (_RB10!=2 && _RC10!=2) {
            if (_RB10 == _RC10) {
                A5LZQueue.clkCondA = true;
                A5LZQueue.clkCondD = true;
            }
            else {
                A5LZQueue.clkCondB = true;
                A5LZQueue.clkCondC = true;
            }
        }
        else {
            A5LZQueue.clkCondA = true;
            A5LZQueue.clkCondB = true;
            A5LZQueue.clkCondC = true;
            A5LZQueue.clkCondD = true;
        }
        A5LZQueue.clkCondArray = &A5LZQueue.dctItemPointer->clkCondArray[0];

        A5LZQueue.clkCondArray[0] = A5LZQueue.clkCondA;
        A5LZQueue.clkCondArray[1] = A5LZQueue.clkCondB;
        A5LZQueue.clkCondArray[2] = A5LZQueue.clkCondC;
        A5LZQueue.clkCondArray[3] = A5LZQueue.clkCondD;
    }



static __device__ __host__  void AddNewItem(A5LogicalZonotopeQueue& A5LZQueue, int& countCurr,  int& RAind, int& RBind, int& RCind) {
        A5LZQueue.stackIndex++;
        // A5LZQueue.dctItem = A5LZQueue._stack[A5LZQueue.stackIndex];
        A5LZQueue.dctItemPointer = &A5LZQueue._stack[A5LZQueue.stackIndex];

        A5LZQueue.dctItemPointer->countCurr = countCurr;
        A5LZQueue.dctItemPointer->RAind = RAind;
        A5LZQueue.dctItemPointer->RBind = RBind;
        A5LZQueue.dctItemPointer->RCind = RCind;

        //  A5LZQueue.clkCondArray = A5LZQueue.dctItem.clkCondArray;
        MajorityFunction(A5LZQueue, RAind, RBind, RCind);
    }



__device__ __host__ static void MajorityFunction(uint8_t& RA8, uint8_t& RB10, uint8_t& RC10, bool(&clkCondArray)[4])
{
    clkCondArray[0] = false; clkCondArray[1] = false; clkCondArray[2] = false; clkCondArray[3] = false;

    // Check if RA9, RB11, RC11 generator property is empty or not
    if (RA8!=2 && RB10!=2 && RC10!=2)
    {
        // If empty, then check if RA9, RB11, RC11 points are equal or not
        if (RA8 == RB10&& RB10== RC10) {
            clkCondArray[0] = true;
        }
        else if (RA8== RB10) {
            clkCondArray[1] = true;
        }
        else if (RA8== RC10) {
            clkCondArray[2] = true;
        }
        else if (RB10== RC10) {
            clkCondArray[3] = true;
        }
    }
    else if (RA8!=2 && RB10!=2) {
        if (RA8== RB10) {
            clkCondArray[0] = true;
            clkCondArray[1] = true;
        }
        else {
            clkCondArray[2] = true;
            clkCondArray[3] = true;
        }
    }
    else if (RA8!=2 && RC10!=2) {
        if (RA8 == RC10) {
            clkCondArray[0] = true;
            clkCondArray[2] = true;
        }
        else {
            clkCondArray[1] = true;
            clkCondArray[3] = true;
        }
    }
    // Check the RC11 and RB11 generators
    else if (RB10!=2 && RC10!=2) {
        if (RB10 == RC10) {
            clkCondArray[0] = true;
            clkCondArray[3] = true;
        }
        else {
            clkCondArray[1] = true;
            clkCondArray[2] = true;
        }
    }
    else {
        clkCondArray[0] = true;
        clkCondArray[1] = true;
        clkCondArray[2] = true;
        clkCondArray[3] = true;
    }



}

    // TODO: create a Destructor for the  struct
    //~A5LogicalZonotopeQueue() {
    //    for (int i = 0; i < _count; i++) {
    //        delete[] std::get<4>(_stack[i]);  // Free the bool array
    //    }
    //    delete[]  A5LZQueue._tempRA;  // Free the uint8_t arrays
    //    delete[]  A5LZQueue._tempRB;
    //    delete[]  A5LZQueue._tempRC;
    // 
    // 
    //}



