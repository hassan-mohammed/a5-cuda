#include "hip/hip_runtime.h"
﻿#define MultiCard true 
#include "hip/hip_runtime_api.h"
#include <time.h>
#include <iostream>
#include <vector>
#include "TestCases.cpp"
#include "Helper.cpp"
#include <omp.h>
#include <tbb/concurrent_vector.h>
#include ""
#include <stdio.h>
#include "A5Breaker_LogicalZonotope.cu"
#include "A5-1-Breaker.h"

#pragma once

bool useTestingKey;
bool useKnownRandom;
int RAkey[RAlength];
int RBkey[RBlength];
int RCkey[RClength];


__device__  __constant__ size_t perThreadMemorySize = (d_RAlength + d_RBlength + d_RClength) * sizeof(uint8_t);

__constant__ __device__   int threeBitsTableLength = 8;
 __device__   bool isKeyFound = false;



__global__ void  FindValidCompinationsGPU  (uint8_t* d_AssumedBitstruthTableZonotope,int** outResult, int * outResultVector, int* d_outStream, int d_count)
{
    //int L = static_cast<int>(std::pow(2, noAssumedBits));
    int itdx = blockIdx.x * blockDim.x + threadIdx.x;
   // uint8_t tempPoint = uint8_t{ 0, 1 };
    // int i = 0, j = 0, k = 1;

    int i = itdx & 0x1F;
    int j = (itdx >> 5) & 0x1F;
    int k = (itdx >> 10) & 0x1F;

    uint8_t tempRAtask[d_RAlength];
    uint8_t tempRBtask[d_RBlength];
    uint8_t tempRCtask[d_RClength];

    GenerateLogicalZonotopeRegisterDevice(tempRAtask, d_RAlength);
    GenerateLogicalZonotopeRegisterDevice(tempRBtask, d_RBlength);
    GenerateLogicalZonotopeRegisterDevice(tempRCtask, d_RClength);

    // Fill last noBits elements of tempRAtask with a row from AssumedBitstruthTableZonotope
    FillLastNBitsWithRowDevice(tempRAtask, d_RAlength, d_AssumedBitstruthTableZonotope, i, d_noAssumedBits);
    FillLastNBitsWithRowDevice(tempRBtask, d_RBlength, d_AssumedBitstruthTableZonotope, j, d_noAssumedBits);
    FillLastNBitsWithRowDevice(tempRCtask, d_RClength, d_AssumedBitstruthTableZonotope, k, d_noAssumedBits);
   /* for (size_t index = 0; index < d_noAssumedBits; index++)
    {
        tempRAtask[d_RAlength - index].Point = (i & (1 << index)) ? 1 : 0;
        tempRBtask[d_RBlength - index].Point = (j & (1 << index)) ? 1 : 0;
        tempRCtask[d_RClength - index].Point = (k & (1 << index)) ? 1 : 0;
        tempRAtask[d_RAlength - index].Generator = 0;
        tempRBtask[d_RBlength - index].Generator = 0;
        tempRCtask[d_RClength - index].Generator = 0;

    }*/
    // PrintRegistersDevice(tempRAtask, tempRBtask, tempRCtask);
     // Create A5LogicalZonotopeQueue with temporary arrays
    A5LogicalZonotopeQueue a5;
     InitializeA5LogicalZonotopeQueue(&a5, tempRAtask, tempRBtask, tempRCtask, d_outStream, d_count);

    bool isValid = false;
    // Explore all clocking branches as the clocking bits are uncertain
    isValid = IsValidKey(a5, 'A');
    if (!isValid) isValid = IsValidKey(a5, 'B');
    if (!isValid) isValid = IsValidKey(a5, 'C');
    if (!isValid) isValid = IsValidKey(a5, 'D');

    if (isValid) {
       // printf("{%d} valid com found {%d},{%d}{%d} \n", itdx, i, j, k);

        int ptr[3] = { i, j, k };
        outResult[itdx] = ptr;

        outResultVector[3 * itdx] = i;
        outResultVector[3 * itdx +1] = j;
        outResultVector[3 * itdx +2] = k;


        //outResult[3 * itdx] = i;
        //outResult[3 * itdx + 1] = j;
        //outResult[3 * itdx + 2] = k;

    }
    else
    {
      // printf("{%d} com is not valid {%d},{%d}{%d} \n", itdx, i, j, k);

        outResult[itdx] = NULL;

        outResultVector[3 * itdx] = -1;
        outResultVector[3 * itdx + 1] = -1;
        outResultVector[3 * itdx + 2] = -1;

        /*outResult[3 * itdx] = NULL;
        outResult[3 * itdx + 1] = NULL;
        outResult[3 * itdx + 2] = NULL;*/
    }

}


__global__ void FindA5Key(const int* __restrict__ outResultVector, const uint8_t* __restrict__ AssumedBitstruthTableZonotope,
    const uint8_t* __restrict__ threeBitsTruthTableZonotope, const int* __restrict__ d_outStream, int  d_count)
{
    // printf("\n ******** GPU FindA5Key Started 0 ******** \n");

    int itdx = blockIdx.x * blockDim.x + threadIdx.x;

    int idx = 3 * itdx, jdx = 3 * itdx + 1, kdx = 3 * itdx + 2;



    if (idx >= 10752 || jdx >= 10750 || kdx >= 10752) {
        printf("Thread %d: Index out of bounds (idx: %d, jdx: %d, kdx: %d)\n", itdx, idx, jdx, kdx);

       return;
    }
    int i = outResultVector[idx];
    int j = outResultVector[jdx];
    int k = outResultVector[kdx];
    printf("\n ******** This is combination is i=%d  j=%d  k=%d ******** \n", i, j, k);

    uint8_t threeBitsLocal[3 * 8];
    for (size_t i = 0; i < 3*8; i++)
    {
        threeBitsLocal[i] = threeBitsTruthTableZonotope[i];

    }

 /*   if ( kdx >= 43000) {
        printf("Thread %d: Index out of bounds (idx: %d, jdx: %d, kdx: %d)\n", itdx, idx, jdx, kdx);
        return;
    }*/



    // printf("\n ******** GPU FindA5Key Started 1 ******** \n");


 /*   if (i == 0 && j == 0)
    {
        printf("\n ******** This is might be a valid combination k=%d ******** \n", k);
    }*/

  
    //// Calculate memory offset for this thread
    //uint8_t* RAcurr = &d_memory[itdx * perThreadMemorySize / sizeof(uint8_t)];
    //uint8_t* RBcurr = RAcurr + RAlength;
    //uint8_t* RCcurr = RBcurr + RBlength;

  /*  extern __shared__ uint8_t sharedMem[];
    uint8_t* RAcurr = sharedMem;
    uint8_t* RBcurr = &sharedMem[d_RAlength];
    uint8_t* RCcurr = &sharedMem[d_RAlength + d_RBlength];*/


    uint8_t RAcurr[d_RAlength];
    uint8_t RBcurr[d_RBlength];
    uint8_t RCcurr[d_RClength];

  //  uint8_t  point = { 0,1 };
    GenerateLogicalZonotopeRegisterDevice(RAcurr, d_RAlength);
    GenerateLogicalZonotopeRegisterDevice(RBcurr, d_RBlength);
    GenerateLogicalZonotopeRegisterDevice(RCcurr, d_RClength);

    /*  printf("\n*** Printing initial values  \n ");
      PrintRegistersDevice(tempRAtask, tempRBtask, tempRCtask);*/


    ////  // Assuming FillLastNBitsWithRow is a function to fill the last N bits with a row from the truthTable
    //FillLastNBitsWithRowDevice(RAcurr, d_RAlength, AssumedBitstruthTableZonotope, i, d_noAssumedBits);
    //FillLastNBitsWithRowDevice(RBcurr, d_RBlength, AssumedBitstruthTableZonotope, j, d_noAssumedBits);
    //FillLastNBitsWithRowDevice(RCcurr, d_RClength, AssumedBitstruthTableZonotope, k, d_noAssumedBits);

    //this one is working fine 
    for (size_t index = 0; index < d_noAssumedBits; index++)
    {
        RAcurr[d_RAlength - d_noAssumedBits + index] = AssumedBitstruthTableZonotope[i * d_noAssumedBits + index];
        RBcurr[d_RBlength - d_noAssumedBits + index] = AssumedBitstruthTableZonotope[j * d_noAssumedBits + index];
        RCcurr[d_RClength - d_noAssumedBits + index] = AssumedBitstruthTableZonotope[k * d_noAssumedBits + index];

    }

    //for (size_t index = 0; index < d_noAssumedBits; index++)
    //{
    //      RAcurr[d_RAlength - d_noAssumedBits + index].Point = (i* d_noAssumedBits & (1 << index)) ? 1 : 0;
    //      RBcurr[d_RBlength - d_noAssumedBits + index].Point = (j* d_noAssumedBits & (1 << index)) ? 1 : 0;
    //      RCcurr[d_RClength - d_noAssumedBits + index].Point = (k* d_noAssumedBits & (1 << index)) ? 1 : 0;
    //      RAcurr[d_RAlength - d_noAssumedBits + index].Generator = 0;
    //      RBcurr[d_RBlength - d_noAssumedBits + index].Generator = 0;
    //      RCcurr[d_RClength - d_noAssumedBits + index].Generator = 0;

    //}
  

    /*  printf("\n*** Printing registed filled with valid combination  \n ");
      PrintRegistersDevice(tempRAtask, tempRBtask, tempRCtask);*/

    int relativeIndex = d_RAlength - d_noAssumedBits + 3;
    // Helper::PrintRegisters(tempRAtask, tempRBtask, tempRCtask);

    // //finalStep(tempRAtask, tempRBtask, tempRCtask, RCInitialIndex);
    //uint8_t* RAcurr = tempRAtask;
    //uint8_t* RBcurr = tempRBtask;
    //uint8_t* RCcurr = tempRCtask;


    // std::string outputString;
    A5LogicalZonotopeQueue A5Zonotope;
    InitializeA5LogicalZonotopeQueue(&A5Zonotope,  RAcurr, RBcurr, RCcurr, d_outStream, d_count);
    //A5PolyLogicalZonotope A5PolyZonotope(RAcurr, RBcurr, RCcurr);

    //ReverseQueue indexQueue(RClength);
    ReverseQueue indexQueue;
    initializeReverseQueue(&indexQueue);

    /*  if (RAcurr[4].Point == 1 && RAcurr[5].Point == 0 && RBcurr[7].Point == 0) {
          outputString = " [Final step]  RC[8]&[9] = " + std::to_string(RCcurr[8].Point) + ',' + std::to_string(RCcurr[9].Point) + " @ " + Helper::GetCurrentTime();
          std::cout << outputString << std::endl;
      }*/
    int index = 0;
    bool isValid;
    bool clkCondArray[4] = { false, false, false, false };
    while (true)
    {
       

       /* if (iterations > 200000);
        break;*/

        while (index < threeBitsTableLength)
        {
           // testing code 
            /*   if (i == 0 && j == 0 && k == 1)
            {
                if (relativeIndex == 17)
                    index = 1;
                else if (relativeIndex == 16)
                    index = 0;
                else if (relativeIndex == 15)
                {
                    // printf("\nrelativeIndex = 15");
                    index = 1;
                }
                else if (relativeIndex == 14)
                {
                    // printf("\nrelativeIndex = 14");
                    index = 2;
                }
                else if (relativeIndex == 13)
                    index = 3;
                else if (relativeIndex == 12)
                    index = 1;
             else if (relativeIndex == 11)
                    index = 4;

            }*/

            // if the relativeIndex is less than 4 then we are working on RB & RC only 
            if (relativeIndex < 4)
            {
                if (relativeIndex == 0 && index > 1) // two values for RC 0 or 1 are tested now. so let's break
                    break;
                if (index > 3)  // all four values for RC and RB are tested now. so let's break
                    break;
            }
            else
                RAcurr[relativeIndex - 4] = threeBitsTruthTableZonotope[index * 3 + 0];//[0];
            if (relativeIndex > 0)
                RBcurr[relativeIndex - 1] = threeBitsTruthTableZonotope[index * 3 + 1];// [1] ;
            RCcurr[relativeIndex] = threeBitsTruthTableZonotope[index * 3 + 2];// [2] ;

            /* printf("\n*** Printing registed filled with three bits more   \n ");
             PrintRegistersDevice(RAcurr, RBcurr, RCcurr);*/
             // Helper::PrintRegisters(RAcurr, RBcurr, RCcurr);

            isValid = false;
            /*    if (isExactPoly)
                {
                    isValid = A5PolyZonotope.IsValidKey();
                }
                else
                {*/

                //TODO: the Majority Function can be skipped at the first few iterations
            MajorityFunction(RAcurr[8], RBcurr[10], RCcurr[10], clkCondArray);
            //   printf("\n***  MajorityFunction success \n ");

            if (clkCondArray[0])
            {
                isValid = IsValidKey(A5Zonotope, 'A');
            }
            if (clkCondArray[1] && !isValid)
                isValid = IsValidKey(A5Zonotope, 'B');
            if (clkCondArray[2] && !isValid)
                isValid = IsValidKey(A5Zonotope, 'C');
            if (clkCondArray[3] && !isValid)
                isValid = IsValidKey(A5Zonotope, 'D');
            // }
            if (isValid)
            {
                if (relativeIndex == 0)
                {
                    isKeyFound = true;
                    printf("\n*** We found a Key \n");
                    PrintRegistersDevice(RAcurr, RBcurr, RCcurr);

                    //std::cout << "\n*** We found a Key @ " << Helper::GetCurrentTime() << "***\n";
                   // Helper::PrintRegisters(RAcurr, RBcurr, RCcurr);
                    return;
                }
                relativeIndex--;
                // indexQueue.Enqueue(index);
                Enqueue(indexQueue, index);
                index = 0;
            }
            else
                index++;
        }
        if (isKeyFound)
            return;
        if (QueueSize(indexQueue) != 0)
        {
                RCcurr[relativeIndex] = 2;
                // index = indexQueue.Dequeue() + 1;
                index = Dequeue(indexQueue) + 1;

                if (relativeIndex > 0)
                {
                    RBcurr[relativeIndex - 1] = 2;
                    if (relativeIndex > 3)
                        RAcurr[relativeIndex - 4] = 2;
                }
                relativeIndex++;
        }
        else
        {
            printf("\n thread %d finished i=%d  j=%d  k=%d ******** \n", itdx, i, j, k);
            return;
        }
       
       // printf("\n thread %d finished \n", itdx);

    }
    printf("\n thread %d finished \n", itdx);





  //  __syncthreads();

    /*

      if (isMixedMode)
          key = A5Breaker_ExactPoly_A5RFBZT_12_DeepModeLoop_A5Loop(tempRAtask.data(), tempRBtask.data(), tempRCtask.data(), indexText);
      else
          key = A5Breaker_LogicalZonotope_A5RFBZT_12_DeepModeLoop_A5Loop(tempRAtask.data(), tempRBtask.data(), tempRCtask.data(), indexText);
          */

          /* std::cout << "\n[Main Thread] " << indexText << " THREAD FINISHED @  " << Helper::GetCurrentTime() << std::endl;

           delete[] tempRAtask;
           delete[] tempRBtask;
           delete[] tempRCtask;

       }});*/



}



__global__ void test_finalstep(int* validGuess, uint8_t* AssumedThTbl, uint8_t* threeThTbl, int* d_outStream, int d_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d, \n", idx);

    printf("\n GPU Assumed BITS  \n");
    for (int i = 0; i < 160; i++)
    {
        printf("{%d,%d} ,", AssumedThTbl[i], AssumedThTbl[i]);

    }

    printf("\n GPU Three BITS  \n");
    for (int i = 0; i < 24; i++)
    {
        printf("{%d,%d} ,", threeThTbl[i], threeThTbl[i]);

    }

    printf("\n GPU VALID Guess BITS  \n");
    for (int i = 0; i < 100; i++)
    {
        printf("{%d, %d, %d}\n ,", validGuess[3 * i], validGuess[3 * i + 1], validGuess[3 * i + 2]);

    }
    printf("\n GPU count = %d  \n", d_count);

    printf("\n GPU VALID Out Stream  \n");
    printf("\n out stream ={");

    for (int i = 0; i < d_count; i++)
    {
        printf("%d, ", d_outStream[i]);

    }
    printf("}");




    /* uint8_t* RA = GenerateLogicalZonotopeRegister(d_RAlength);
     uint8_t* RB = GenerateLogicalZonotopeRegister(d_RBlength);
     uint8_t* RC = GenerateLogicalZonotopeRegister(d_RClength);

     FillLastNBitsWithRow(RA, d_RAlength, AssumedThTbl, 0, d_noAssumedBits);
     FillLastNBitsWithRow(RB, d_RBlength, AssumedThTbl, 0, d_noAssumedBits);
     FillLastNBitsWithRow(RC, d_RClength, AssumedThTbl, 1, d_noAssumedBits);*/


     /*   printf("\n GPU Content of RA,RB,RC Registers  \n");

        printRegisters(RA, RB, RC);*/

}





// Parse the configuration file and generate the initial Key Stream. 
void Initialization()
{

    Helper::ParseConfigFile("./App.config", RAkey, RBkey, RCkey);

    std::cout << "\n**** Attacking Paramters ****\n";
    std::cout << "Key stream length  " << count << std::endl;
    std::cout << "No assumed bits  " << noAssumedBits << std::endl;
    std::cout << "2nd level assumed bits  " << deepNoAssumedBits << std::endl;

    std::cout << "\n**** Secret Key  ****\n";
    Helper::PrintRegisters(RAkey, RBkey, RCkey);
    outStream = A5_1::GenerateSequence(RAkey, RBkey, RCkey, count);

    std::cout << "\n**** Generated Key Stream **** " << std::endl;
    for (size_t i = 0; i < count; i++)
    {
        std::cout << outStream[i] << ",";
    }
    std::cout << std::endl;



}



// Parallel simulation logic
void FindValidCompinations(std::vector<int*>& validGuessConBag, int noAssumedBits)
{
    int L = static_cast<int>(std::pow(2, noAssumedBits));
    static uint8_t* AssumedBitstruthTableZonotope = Helper::GetTruthTableZonotope(noAssumedBits);


    uint8_t RAinit[RAlength];
    uint8_t RBinit[RBlength];
    uint8_t RCinit[RClength];

    Helper::GenerateLogicalZonotopeRegister(RAinit, RAlength);
    Helper::GenerateLogicalZonotopeRegister(RBinit, RBlength);
    Helper::GenerateLogicalZonotopeRegister(RCinit, RClength);

    // Parallel simulation loop
  //  std::vector<std::vector<int*>> privateBags(Concurrency::GetProcessorCount()); // Private bags for each thread


    tbb::concurrent_vector<std::vector<int*>> privateBags(tbb::this_task_arena::max_concurrency());

    // #pragma omp parallel for
    //for (int index = 0; index < L; index++) {
    tbb::parallel_for(0, L, [&](int index) {
        // Create temporary copies of RA, RB, RC
    /*    uint8_t tempRAtask[RAlength];
        uint8_t tempRBtask[RAlength];
        uint8_t tempRCtask[RAlength];*/

        uint8_t tempRAtask[RAlength];
        uint8_t tempRBtask[RBlength];
        uint8_t tempRCtask[RClength];

        // Copy values from RA, RB, RC to temporary arrays
        std::copy(RAinit, RAinit + RAlength, tempRAtask);
        std::copy(RBinit, RBinit + RBlength, tempRBtask);
        std::copy(RCinit, RCinit + RClength, tempRCtask);

        // Fill last noBits elements of tempRAtask with a row from AssumedBitstruthTableZonotope
        Helper::FillLastNBitsWithRow(tempRAtask, RAlength, AssumedBitstruthTableZonotope, index, noAssumedBits);
        A5LogicalZonotopeQueue a5;
        // Create A5LogicalZonotopeQueue with temporary arrays
         InitializeA5LogicalZonotopeQueue(&a5, tempRAtask, tempRBtask, tempRCtask, outStream, count);
        //  A5PolyLogicalZonotope A5PolyZonotope(tempRAtask, tempRBtask, tempRCtask);


          // Nested loops for RB and RC
        for (int j = 0; j < L; j++) {
            Helper::FillLastNBitsWithRow(tempRBtask, RBlength, AssumedBitstruthTableZonotope, j, noAssumedBits);

            for (int k = 0; k < L; k++) {
                Helper::FillLastNBitsWithRow(tempRCtask, RClength, AssumedBitstruthTableZonotope, k, noAssumedBits);
                bool isValid = false;

                //Helper::PrintRegisters(tempRAtask, tempRBtask, tempRCtask);
                if (isExactPoly)
                {
                    // isValid = A5PolyZonotope.IsValidKey();
                }
                else
                {
                    // Explore all clocking branches as the clocking bits are uncertain
                    isValid = IsValidKey(a5, 'A');
                    if (!isValid) isValid = IsValidKey(a5, 'B');
                    if (!isValid) isValid = IsValidKey(a5, 'C');
                    if (!isValid) isValid = IsValidKey(a5, 'D');
                    //    std::cout << "is valid = " << std::boolalpha << isValid << "\n" << std::endl;
                }
                // If key is valid, add the combination to the validGuessConBag
                if (isValid) {
                    int* ptr = new int[3] { index, j, k };

                    // int ptr[3];
                    /* ptr[0] = index;
                     ptr[1] = j;
                     ptr[2] = k;*/

                     // validGuessConBag.push_back(ptr);

                    privateBags[tbb::this_task_arena::current_thread_index()].push_back(ptr);
                }
            }
        }

  
        });

    // Merge private bags into the main bag
    for (auto& privateBag : privateBags) {
        validGuessConBag.insert(validGuessConBag.end(), privateBag.begin(), privateBag.end());
    }
    privateBags.clear();

    //for (int i = 0; i < omp_get_num_threads(); i++) {
    //    validGuessConBag.insert(validGuessConBag.end(), privateBags[i].begin(), privateBags[i].end());
    //    privateBags[i].clear();  // Clear the private bag after merging
    //}

    // Merge private bags into the main bag
   /* for (auto& privateBag : privateBags) {
        validGuessConBag.insert(validGuessConBag.end(), privateBag.begin(), privateBag.end());
    }*/



}







int main()
{
    hipFree(0);
    std::cout << "==============================================================\n";
    std::cout << "             A5 Breaker using logicalZonotope !\n";
    std::cout << "==============================================================\n";


    //RunTestCases();
    //Helper::A5StreamCalcultionTime(100);
   // DeviceProperties();
    //
    
      // Create a thread that runs the printCurrentDateTime function
  //  std::thread backgroundThread(Helper::printCurrentDateTime);

    // Detach the thread so it runs independently
  //  backgroundThread.detach();

    Initialization();
  
    std::vector<int*> validGuessConBag;
    FindValidCompinationsCPU(validGuessConBag);

          //std::for_each( validGuessConBag.begin(), validGuessConBag.end(), [&](int* index) {
      //#pragma omp parallel for num_threads(8)
      //for (int it = 0; it < static_cast<int>(validGuessConBag.size()); ++it) {
      //    int* index = validGuessConBag[it];
      //    int i = index[0], j = index[1], k = index[2];


    //lenght is defined by (the raws of truth table * no bits) because I have all in one dimension array 
    int validGuessVectorSize = validGuessConBag.size() * 3;
    int* validGuessVector = new int[validGuessVectorSize];

    for (int i = 0; i < validGuessConBag.size(); i++)
    {
        validGuessVector[3 * i] = validGuessConBag[i][0];
        validGuessVector[3 * i + 1] = validGuessConBag[i][1];
        validGuessVector[3 * i + 2] = validGuessConBag[i][2];
    }


   /* printf("\n CPU Assumed BITS  \n");
    for (int i = 0; i < assumedBitsTruthTblLen; i++)
    {
        printf("{%d,%d} ,", AssumedBitstruthTableZonotope[i].Point, AssumedBitstruthTableZonotope[i].Generator);

    }

    printf("\n CPU Three BITS  \n");
    for (int i = 0; i < threeBitsTruthTblLen; i++)
    {
        printf("{%d,%d} ,", threeBitsTruthTableZonotope[i].Point, threeBitsTruthTableZonotope[i].Generator);

    }

    printf("\n CPU VALID Guess BITS  \n");
    for (int i = 0; i < 100; i++)
    {
        printf("{%d, %d, %d}\n ,", validGuessConBag[i][0], validGuessConBag[i][1], validGuessConBag[i][2]);

    }*/

    /*******  Cuda work section *******/

    std::cout << " \n======== GPU Section: =========== " << std::endl;
    int numDevices;
    checkCudaErrors(hipGetDeviceCount(&numDevices));

    std::cout << "******* No of GPU Cards is: " << numDevices << " *******" << std::endl;


    uint8_t* d_threeBitsTruthTableZonotope = NULL;
    int* d_validGuessConBag = NULL;
    int* d_outStream = NULL;
    uint8_t* d_AssumedBitstruthTableZonotope = NULL;
    uint8_t* AssumedBitstruthTableZonotope = Helper::GetTruthTableZonotope(noAssumedBits);

    int partSize = validGuessVectorSize / 4;
    int* d_VectorParts[4];
    if (MultiCard)
    {
        for (size_t i = 0; i < numDevices; i++)
        {
            checkCudaErrors(hipSetDevice(i));
            checkCudaErrors(hipMalloc((void**)&d_VectorParts[i], partSize * sizeof(int)));
            checkCudaErrors(hipMemcpy(d_VectorParts[i], validGuessVector + i * partSize, partSize * sizeof(int), hipMemcpyHostToDevice));
        }
    }
  
    AllocateGPUMemory(d_outStream, d_AssumedBitstruthTableZonotope,AssumedBitstruthTableZonotope, d_threeBitsTruthTableZonotope,
        d_validGuessConBag, validGuessVectorSize, validGuessVector);

    /*******  Find valid combinations section *******/
    /*std::vector<int> validItemsVector;*/
    LaunchFindValidCompinationsKernel(d_AssumedBitstruthTableZonotope, d_outStream);
    /*int validItemsVectorSize = validItemsVector.size();
    std::sort(validGuessConBag.begin(), validGuessConBag.end(), Helper::compareIntArrays);*/

    /******* FindA5Key Section *******/
 /*   int numThreads = 5;
    hipError_t cuda_err;

    cuda_err = hipMalloc((void**)&d_validGuessConBag, validItemsVectorSize * sizeof(int));
    if (cuda_err != hipSuccess) {
        std::cout << "Error Allocating the d_validGuessConBag.\n";
    }

    cuda_err = hipMemcpy(d_validGuessConBag, validItemsVector.data(), validItemsVectorSize * sizeof(int), hipMemcpyHostToDevice);
    if (cuda_err != hipSuccess) {
        std::cout << "Error Copying the d_validGuessConBag.\n";
    }*/

 /*   size_t sharedMemSize = (d_RAlength + d_RBlength + d_RClength) * sizeof(uint8_t);
    size_t totalMemorySize = numThreads * (RAlength + RBlength + RClength) * sizeof(uint8_t);
    uint8_t* d_memory;
    hipMalloc(&d_memory, totalMemorySize);*/
    
    std::cout << "\n[GPU call] FindA5Key " << " Started  @  " << Helper::GetCurrentTime() << std::endl;

    // Launch kernels on each GPU
    int threadsPerBlock = 256;
    int blocksPerGrid = (partSize + threadsPerBlock * 3 - 1) / (threadsPerBlock * 3);

    if (MultiCard)
    {
        int i = 0;
            checkCudaErrors(hipSetDevice(i));
            FindA5Key << <14, 256 >> > (d_validGuessConBag, d_AssumedBitstruthTableZonotope, d_threeBitsTruthTableZonotope, d_outStream, count);
    }
    else
    {
        //FindA5Key << <112, 128 >> > (d_validGuessConBag, d_AssumedBitstruthTableZonotope, d_threeBitsTruthTableZonotope, d_outStream, count);
        FindA5Key << <112, 128 >> > (d_validGuessConBag, d_AssumedBitstruthTableZonotope, d_threeBitsTruthTableZonotope, d_outStream, count);
        hipDeviceSynchronize();
        // FindA5Key(validGuessConBag, AssumedBitstruthTableZonotope, threeBitsTruthTableZonotope);

    }


  

    std::cout << "\n[GPU call] FindA5Key " << " Finished  @  " << Helper::GetCurrentTime() << std::endl;




    /* uint8_t* d_tempRA = NULL;
     uint8_t* d_tempRB = NULL;
     uint8_t* d_tempRC = NULL;

     cuda_err = hipMalloc((void**)&d_tempRA, sizeof(uint8_t) * RAlength);
     if (cuda_err != hipSuccess) {
         std::cout << "Error Allocating the d_tempRA.\n";
     }
     cuda_err = hipMalloc((void**)&d_tempRB, sizeof(uint8_t) * RBlength);
     if (cuda_err != hipSuccess) {
         std::cout << "Error Allocating the RB.\n";
     }
     cuda_err = hipMalloc((void**)&d_tempRC, sizeof(uint8_t) * RClength);
     if (cuda_err != hipSuccess) {
         std::cout << "Error Allocating the RC.\n";
     }*/

     //// Copy the data to GPU
     //cuda_err = hipMemcpy(d_tempRA, tempRAtask, sizeof(uint8_t) * RAlength, hipMemcpyHostToDevice);
     //if (cuda_err != hipSuccess) {
     //    std::cout << "Error Copying the RA.\n";
     //}
     //cuda_err = hipMemcpy(d_tempRB, tempRBtask, sizeof(uint8_t) * RBlength, hipMemcpyHostToDevice);
     //if (cuda_err != hipSuccess) {
     //    std::cout << "Error Copying the RA.\n";
     //}
     //cuda_err = hipMemcpy(d_tempRC, tempRCtask, sizeof(uint8_t) * RClength, hipMemcpyHostToDevice);
     //if (cuda_err != hipSuccess) {
     //    std::cout << "Error Copying the RA.\n";
     //}



     /*

       if (isMixedMode)
           key = A5Breaker_ExactPoly_A5RFBZT_12_DeepModeLoop_A5Loop(tempRAtask.data(), tempRBtask.data(), tempRCtask.data(), indexText);
       else
           key = A5Breaker_LogicalZonotope_A5RFBZT_12_DeepModeLoop_A5Loop(tempRAtask.data(), tempRBtask.data(), tempRCtask.data(), indexText);
           */

           // std::cout << "\n[Main Thread] " << indexText << " THREAD FINISHED @  " << Helper::GetCurrentTime() << std::endl;

          /*  delete[] tempRAtask;
            delete[] tempRBtask;
            delete[] tempRCtask;*/

            // }});




        // A5LogicalZonotopeQueue::_outStream = &outStream;
         //A5LogicalZonotopeQueue::_count = count;


    return 0;
}


void LaunchFindValidCompinationsKernel( uint8_t* d_AssumedBitstruthTableZonotope, int* d_outStream)
{
    std::vector<int> validItemsVector;
    int* outResultVector;
    int noPossibilities = static_cast<int>(std::pow(2, noAssumedBits));
    int totalIterations = noPossibilities * noPossibilities * noPossibilities;

    int** outResult; // Device pointer to store output
    // Allocate memory on the device to store the output
    hipMalloc((void**)&outResult, totalIterations * sizeof(int*));
    hipMalloc((void**)&outResultVector,3* totalIterations * sizeof(int));


    std::cout << "\n[GPU call] Find ValidCompinations " << " Started  @  " << Helper::GetCurrentTime() << std::endl;
    FindValidCompinationsGPU << <noPossibilities * noPossibilities, noPossibilities >> > (d_AssumedBitstruthTableZonotope, outResult, outResultVector, d_outStream, count);
    //FindValidCompinationsGPU << <1, 128 >> > (outResult);
    std::cout << "\n[GPU call] Find ValidCompinations " << " Finished  @  " << Helper::GetCurrentTime() << std::endl;

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    hipDeviceSynchronize();

    std::vector<int*> outResultvalidGuessConBag(totalIterations);
    hipMemcpy(outResultvalidGuessConBag.data(), outResult, totalIterations * sizeof(int*), hipMemcpyDeviceToHost);

    std::vector<int*> validItems;
    for (int i = 0; i < outResultvalidGuessConBag.size(); i++) {
        if (outResultvalidGuessConBag[i] != nullptr) {
            // std::cout << outResultvalidGuessConBag[i][0] << ", " << outResultvalidGuessConBag[i][1] << ", " << outResultvalidGuessConBag[i][2] << std::endl;
            validItems.push_back(outResultvalidGuessConBag[i]); // Push valid items to the new vector
        }
    }
    std::cout << "Total number of valid items: " << validItems.size() << std::endl;


    std::vector<int> h_outResultVector(3*totalIterations);
    hipMemcpy(h_outResultVector.data(), outResultVector, 3* totalIterations * sizeof(int), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < h_outResultVector.size(); i++) {
        if (h_outResultVector[i] != -1) {
            // std::cout << outResultvalidGuessConBag[i][0] << ", " << outResultvalidGuessConBag[i][1] << ", " << outResultvalidGuessConBag[i][2] << std::endl;
            validItemsVector.push_back(h_outResultVector[i]); // Push valid items to the new vector
        }
    }
    std::cout << "Total number of valid items VECTOR: " << validItemsVector.size()/3 << std::endl;
    
//    std::vector<int*> validGuessArrays(validItemsVector.size() / 3);
//    int* tmpGuess;
//    for (size_t i = 0; i < validItemsVector.size() / 3;)
//    {
//        tmpGuess = new int [3]{ validItemsVector[i],validItemsVector[i + 1], validItemsVector[i + 2] };
//        validGuessArrays.push_back(tmpGuess);
//        i = i + 3;
//    }
//
////    std::sort(validGuessArrays.begin(), validGuessArrays.end(), Helper::compareIntArrays);
//
//
//    // Calculate the number of groups
//    int numGroups = validItemsVector.size() / 3;
//
//    // Initialize the indices vector
//    std::vector<int> indices(numGroups);
//    for (int i = 0; i < numGroups; ++i) {
//        indices[i] = i * 3;
//    }
//
//    // Sort the indices based on the comparison of groups
//    std::sort(indices.begin(), indices.end(),
//        [&validItemsVector](int a, int b) {
//            return Helper::compareGroups(validItemsVector, a, b);
//        });
//
//    // Create a new vector for sorted elements
//    std::vector<int> sortedVector(validItemsVector.size());
//    for (size_t i = 0; i < indices.size(); ++i) {
//        sortedVector[i * 3] = validItemsVector[indices[i]];
//        sortedVector[i * 3 + 1] = validItemsVector[indices[i] + 1];
//        sortedVector[i * 3 + 2] = validItemsVector[indices[i] + 2];
//    }
//
//    // Print the sorted results
//    for (size_t i = 0; i < sortedVector.size(); i += 3) {
//        std::cout << sortedVector[i] << " "
//            << sortedVector[i + 1] << " "
//            << sortedVector[i + 2] << std::endl;
//    }
}
//FindValidCompinationsCPU 
void FindValidCompinationsCPU(std::vector<int*>& validGuessConBag)
{

    std::cout << " ======== CPU Section: =========== " << std::endl;

    // Start the timer
    auto start = std::chrono::high_resolution_clock::now();
    FindValidCompinations(validGuessConBag, noAssumedBits);

    // Stop the timer
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "Time taken by function: " << duration.count() << " milliseconds" << std::endl;

    std::cout << "Total Number of Guesses: " << validGuessConBag.size() << std::endl;

   //std::sort(validGuessConBag.begin(), validGuessConBag.end(), Helper::compareIntArrays);

   // for (const auto& guess : validGuessConBag) {
   // std::cout << "Valid Guess: {" << guess[0] << ", " << guess[1] << ", " << guess[2] << "}" << std::endl;
   // }

}


void AllocateGPUMemory(int*& d_outStream, uint8_t*& d_AssumedBitstruthTableZonotope, uint8_t* AssumedBitstruthTableZonotope,
    uint8_t*& d_threeBitsTruthTableZonotope, int*& d_validGuessConBag, int validGuessVectorSize, int* validGuessVector)
{

    int assumedBitsTruthTblLen = static_cast<int>(std::pow(2, noAssumedBits)) * noAssumedBits;
    int threeBitsTruthTblLen = static_cast<int>(std::pow(2, 3)) * 3;
    uint8_t* threeBitsTruthTableZonotope = Helper::GetTruthTableZonotope(1 * 3);

    checkCudaErrors(hipMalloc((void**)&d_outStream, sizeof(int) * count));
    checkCudaErrors(hipMemcpy(d_outStream, outStream, sizeof(int) * count, hipMemcpyHostToDevice));
    
    checkCudaErrors(hipMalloc((void**)&d_AssumedBitstruthTableZonotope, sizeof(uint8_t) * assumedBitsTruthTblLen));
    checkCudaErrors(hipMemcpy(d_AssumedBitstruthTableZonotope, AssumedBitstruthTableZonotope, sizeof(uint8_t) * assumedBitsTruthTblLen, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&d_threeBitsTruthTableZonotope, sizeof(uint8_t) * threeBitsTruthTblLen));
    checkCudaErrors(hipMemcpy(d_threeBitsTruthTableZonotope, threeBitsTruthTableZonotope, sizeof(uint8_t) * threeBitsTruthTblLen, hipMemcpyHostToDevice));


    if (!MultiCard)
    {

        checkCudaErrors(hipMalloc((void**)&d_validGuessConBag, validGuessVectorSize * sizeof(int)));
        checkCudaErrors(hipMemcpy(d_validGuessConBag, validGuessVector, validGuessVectorSize * sizeof(int), hipMemcpyHostToDevice));

    }

}


void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(err);
    }
}

void DeviceProperties()
{
    hipDeviceProp_t devprop;
    hipGetDeviceProperties(&devprop, 0);
    std::cout << "Maximum number of thread: " << devprop.maxThreadsPerBlock << std::endl;
    std::cout << "maxThreadsDim : " << devprop.maxThreadsDim[0] << std::endl;
    std::cout << "Clockrate: " << devprop.clockRate << std::endl;

    std::cout << "Multi proccessor count: " << devprop.multiProcessorCount << std::endl;


    size_t memfree, memtoal;
    hipMemGetInfo(&memfree, &memtoal);
    std::cout << "memory free: " << memfree / (1024 * 1024) << std::endl;
    //std::cout << "memory total: " << memtoal / (1024 * 1024) << std::endl;

    std::cout << "memory free: " << memfree / (1024 * 1024) << std::endl;



}

void RunTestCases()
{
    TestCases::TestAnd();
    TestCases::TestOr();
    TestCases::TestNot();
   //  TestCases::testReverseQueueSt();


    int n = 5;
    // Generate and print the truth table
    int* truthTable = Helper::GetTruthTable(n);
    // Helper::PrintTruthTable(truthTable, n);
     // Deallocate memory for the truth table
    delete[] truthTable;

    // Generate and print the logical zonotope truth table
    uint8_t* truthTableZonotope = Helper::GetTruthTableZonotope(n);
    // Helper::PrintTruthTableZonotope(truthTableZonotope, n);
}
