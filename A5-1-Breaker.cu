
#include <hip/hip_runtime.h>
//// A5-1-Breaker.cpp : This file contains the 'main' function. Program execution begins and ends there.
////
//
//
//#include "cuda_runtime.h"
//#include <iostream>
//#include <vector>
//#include "TestCases.cpp"
//#include "A5Breaker_LogicalZonotope.cpp"
//#include "A5-1-Breaker.h"
//#include "Helper.cpp"
//#include <omp.h>
//#include <tbb/concurrent_vector.h>
//#include "A5LogicalZonotopeQueue.cpp"
//
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//
//#pragma once
////int A5LogicalZonotopeQueue::_count = 0;
////std::vector<int>* A5LogicalZonotopeQueue::_outStream = nullptr;
//
//// Global variables
//bool useTestingKey;
//bool useKnownRandom;
//int RA[RAlength];
//int RB[RBlength];
//int RC[RClength];
//LightLogicalZonotope* AssumedBitstruthTableZonotope;
//
//// Parse the configuration file and generate the initial Key Stream. 
//void Initialization()
//{
//
//    Helper::ParseConfigFile("App.config",RA, RB, RC);
//
//    std::cout << "\n**** Attacking Paramters ****\n";
//    std::cout << "Key stream length  " << count << std::endl;
//    std::cout << "No assumed bits  " << noAssumedBits << std::endl;
//    std::cout << "2nd level assumed bits  " << deepNoAssumedBits << std::endl;
//
//    std::cout << "\n**** Secret Key  ****\n";
//    Helper::PrintRegisters(RA, RB, RC);
//    outStream = A5_1::GenerateSequence(RA, RB, RC, count);
//
//    std::cout << "\n**** Generated Key Stream **** " << std::endl;
//    for (int value : outStream) {
//        std::cout << value << ",";
//    }
//    std::cout << std::endl;
//    AssumedBitstruthTableZonotope = Helper::GetTruthTableZonotope(noAssumedBits);
//
//
//}
//
//
//
//// Parallel simulation logic
//void FindValidCompinations(std::vector<int*>& validGuessConBag, int noAssumedBits)
//{
//    int L = static_cast<int>(std::pow(2, noAssumedBits));
//    LightLogicalZonotope* RA = Helper::GenerateLogicalZonotopeRegister(RAlength);
//    LightLogicalZonotope* RB = Helper::GenerateLogicalZonotopeRegister(RBlength);
//    LightLogicalZonotope* RC = Helper::GenerateLogicalZonotopeRegister(RClength);
//    // Parallel simulation loop
//  //  std::vector<std::vector<int*>> privateBags(Concurrency::GetProcessorCount()); // Private bags for each thread
//
//  
//    tbb::concurrent_vector<std::vector<int*>> privateBags(tbb::this_task_arena::max_concurrency());
//
//    // #pragma omp parallel for
//    //for (int index = 0; index < L; index++) {
//    tbb::parallel_for(0, L, [&](int index) {
//        // Create temporary copies of RA, RB, RC
//    /*    LightLogicalZonotope tempRAtask[RAlength];
//        LightLogicalZonotope tempRBtask[RAlength];
//        LightLogicalZonotope tempRCtask[RAlength];*/
//        LightLogicalZonotope* tempRAtask = new LightLogicalZonotope[RAlength];
//        LightLogicalZonotope* tempRBtask = new LightLogicalZonotope[RBlength];
//        LightLogicalZonotope* tempRCtask = new LightLogicalZonotope[RClength];
//
//        // Copy values from RA, RB, RC to temporary arrays
//        std::copy(RA, RA + RAlength, tempRAtask);
//        std::copy(RB, RB + RBlength, tempRBtask);
//        std::copy(RC, RC + RClength, tempRCtask);
//
//        // Fill last noBits elements of tempRAtask with a row from AssumedBitstruthTableZonotope
//        Helper::FillLastNBitsWithRow(tempRAtask, RAlength, AssumedBitstruthTableZonotope, index, noAssumedBits);
//
//        // Create A5LogicalZonotopeQueue with temporary arrays
//        A5LogicalZonotopeQueue a5 = InitializeA5LogicalZonotopeQueue(tempRAtask, tempRBtask, tempRCtask, &outStream, count);
//      //  A5PolyLogicalZonotope A5PolyZonotope(tempRAtask, tempRBtask, tempRCtask);
//
//
//        // Nested loops for RB and RC
//        for (int j = 0; j < L; j++) {
//            Helper::FillLastNBitsWithRow(tempRBtask, RBlength, AssumedBitstruthTableZonotope, j, noAssumedBits);
//
//            for (int k = 0; k < L; k++) {
//                Helper::FillLastNBitsWithRow(tempRCtask, RClength, AssumedBitstruthTableZonotope, k, noAssumedBits);
//                bool isValid = false;
//
//                //Helper::PrintRegisters(tempRAtask, tempRBtask, tempRCtask);
//                if (isExactPoly)
//                {
//                   // isValid = A5PolyZonotope.IsValidKey();
//                }
//                else
//                {
//                    // Explore all clocking branches as the clocking bits are uncertain
//                    isValid = IsValidKey(a5,'A');
//                    if (!isValid) isValid = IsValidKey(a5,'B');
//                    if (!isValid) isValid = IsValidKey(a5,'C');
//                    if (!isValid) isValid = IsValidKey(a5,'D');
//                    //    std::cout << "is valid = " << std::boolalpha << isValid << "\n" << std::endl;
//                }
//                    // If key is valid, add the combination to the validGuessConBag
//                if (isValid) {
//                    int* ptr = new int[3] { index, j, k };
//
//                    // int ptr[3];
//                    /* ptr[0] = index;
//                     ptr[1] = j;
//                     ptr[2] = k;*/
//
//                     // validGuessConBag.push_back(ptr);
//
//                   privateBags[tbb::this_task_arena::current_thread_index()].push_back(ptr);
//                }
//            }
//        }
//
//        delete[] tempRAtask;
//        delete[] tempRBtask;
//        delete[] tempRCtask;
//        });
//
//    // Merge private bags into the main bag
//    for (auto& privateBag : privateBags) {
//        validGuessConBag.insert(validGuessConBag.end(), privateBag.begin(), privateBag.end());
//    }
//    privateBags.clear();
//
//    //for (int i = 0; i < omp_get_num_threads(); i++) {
//    //    validGuessConBag.insert(validGuessConBag.end(), privateBags[i].begin(), privateBags[i].end());
//    //    privateBags[i].clear();  // Clear the private bag after merging
//    //}
//
//    // Merge private bags into the main bag
//   /* for (auto& privateBag : privateBags) {
//        validGuessConBag.insert(validGuessConBag.end(), privateBag.begin(), privateBag.end());
//    }*/
//
//   
//
//}
//
//

//
//
//int main()
//{
//    int a = 5;
//    
//    cudaDeviceSynchronize();
//        return 0;
//}
//
//
//
//
//
////int main()
////{
////    LightLogicalZonotope* RA = Helper::GenerateLogicalZonotopeRegister(RAlength);
////    LightLogicalZonotope* RB = Helper::GenerateLogicalZonotopeRegister(RBlength);
////    LightLogicalZonotope* RC = Helper::GenerateLogicalZonotopeRegister(RClength);
////
////    std::cout << "==============================================================\n";
////    std::cout << "             A5 Breaker using logicalZonotope !\n";
////    std::cout << "==============================================================\n";
////
////    test1<<<1, 1 >>>();
////    cudaDeviceSynchronize();
////    return 0;
////
////    // **  Running some test Cases
////    RunTestCases();
////    //Helper::A5StreamCalcultionTime(100);
////    //
////    std::vector<int*> validGuessConBag;
////
////    Initialization();
////   // A5LogicalZonotopeQueue::InitializeA5LogicalZonotopeQueue(&outStream, count);
////
////    // Start the timer
////    auto start = std::chrono::high_resolution_clock::now();
////    FindValidCompinations(validGuessConBag, noAssumedBits);
////
////    // Stop the timer
////    auto stop = std::chrono::high_resolution_clock::now();
////    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
////    std::cout << "Time taken by function: " << duration.count() << " milliseconds" << std::endl;
////
////    std::cout << "Total Number of Guesses: " << validGuessConBag.size() << std::endl;
////
////    std::sort(validGuessConBag.begin(), validGuessConBag.end(), Helper::compareIntArrays);
////
////    /*  for (const auto& guess : validGuessConBag) {
////          std::cout << "Valid Guess: {" << guess[0] << ", " << guess[1] << ", " << guess[2] << "}" << std::endl;
////      }*/
////
////
////      /****** Starting the Deep Mode*******/
////
////  //  InitializeStaticMembers();
////
////
////    //std::for_each( validGuessConBag.begin(), validGuessConBag.end(), [&](int* index) {
////    //#pragma omp parallel for num_threads(8)
////    //for (int it = 0; it < static_cast<int>(validGuessConBag.size()); ++it) {
////    //    int* index = validGuessConBag[it];
////    //    int i = index[0], j = index[1], k = index[2];
////
////    if (isExactPoly)
////    {
////        std::cout << std::endl  << "**********Operating in Exact Poly Mode*******" << std::endl;
////    }
////
////        // Use TBB for parallel processing
////  /*  tbb::parallel_for(tbb::blocked_range<size_t>(0, validGuessConBag.size()),
////        [&](const tbb::blocked_range<size_t>& r) {
////            for (size_t it = r.begin(); it != r.end(); ++it) {
////                int* index = validGuessConBag[it];
////                int i = index[0], j = index[1], k = index[2];*/
////    
////               int i = 0, j = 0, k = 1;
////
////                std::string indexText = "i = " + std::to_string(i) + ", j = " + std::to_string(j) + ", k = " + std::to_string(k);
////                std::cout << "\n[Main Thread] " << indexText << " Started  @  " << Helper::GetCurrentTime() << std::endl;
////
////                // Create temporary copies of RA, RB, RC
////                LightLogicalZonotope* tempRAtask = new LightLogicalZonotope[RAlength];
////                LightLogicalZonotope* tempRBtask = new LightLogicalZonotope[RBlength];
////                LightLogicalZonotope* tempRCtask = new LightLogicalZonotope[RClength];
////
////                // Copy values from RA, RB, RC to temporary arrays
////                std::copy(RA, RA + RAlength, tempRAtask);
////                std::copy(RB, RB + RBlength, tempRBtask);
////                std::copy(RC, RC + RClength, tempRCtask);
////                // Assuming FillLastNBitsWithRow is a function to fill the last N bits with a row from the truthTable
////                Helper::FillLastNBitsWithRow(tempRAtask, RAlength, AssumedBitstruthTableZonotope, i, noAssumedBits);
////                Helper::FillLastNBitsWithRow(tempRBtask, RBlength, AssumedBitstruthTableZonotope, j, noAssumedBits);
////                Helper::FillLastNBitsWithRow(tempRCtask, RClength, AssumedBitstruthTableZonotope, k, noAssumedBits);
////
////                bool key = false;
////                int RCInitialIndex = RAlength - noAssumedBits + 3;
////               finalStep(tempRAtask, tempRBtask, tempRCtask, RCInitialIndex);
////
////                /*
////
////                  if (isMixedMode)
////                      key = A5Breaker_ExactPoly_A5RFBZT_12_DeepModeLoop_A5Loop(tempRAtask.data(), tempRBtask.data(), tempRCtask.data(), indexText);
////                  else
////                      key = A5Breaker_LogicalZonotope_A5RFBZT_12_DeepModeLoop_A5Loop(tempRAtask.data(), tempRBtask.data(), tempRCtask.data(), indexText);
////                      */
////
////                std::cout << "\n[Main Thread] " << indexText << " THREAD FINISHED @  "<< Helper::GetCurrentTime() << std::endl;
////               
////                delete[] tempRAtask;
////                delete[] tempRBtask;
////                delete[] tempRCtask;
////
////           // }});
////
////
////
////
////    // A5LogicalZonotopeQueue::_outStream = &outStream;
////     //A5LogicalZonotopeQueue::_count = count;
////
////
////    return 0;
////}
////
////
////
////
////
////void RunTestCases()
////{
////    TestCases::TestAnd();
////    TestCases::TestOr();
////    TestCases::TestNot();
////   TestCases::testReverseQueueSt();
////
////
////    int n = 5;
////    // Generate and print the truth table
////    int* truthTable = Helper::GetTruthTable(n);
////   // Helper::PrintTruthTable(truthTable, n);
////    // Deallocate memory for the truth table
////    delete[] truthTable;
////    
////    // Generate and print the logical zonotope truth table
////    LightLogicalZonotope* truthTableZonotope = Helper::GetTruthTableZonotope(n);
////   // Helper::PrintTruthTableZonotope(truthTableZonotope, n);
////}
